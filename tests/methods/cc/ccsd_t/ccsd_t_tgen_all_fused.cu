#include "hip/hip_runtime.h"
#include "header.hpp"
#include <stdio.h>
#include <stdlib.h>
#include <vector>

// created by tc_gen_definition()
#define FUSION_SIZE_SLICE_1_H3 	4
#define FUSION_SIZE_SLICE_1_H2 	4
#define FUSION_SIZE_SLICE_1_H1 	4
#define FUSION_SIZE_SLICE_1_P6 	4
#define FUSION_SIZE_SLICE_1_P5 	4
#define FUSION_SIZE_SLICE_1_P4 	4
#define FUSION_SIZE_SLICE_1_H7 	16

#define FUSION_SIZE_SLICE_2_H3 	4
#define FUSION_SIZE_SLICE_2_H2 	4
#define FUSION_SIZE_SLICE_2_H1 	4
#define FUSION_SIZE_SLICE_2_P6 	4
#define FUSION_SIZE_SLICE_2_P5 	4
#define FUSION_SIZE_SLICE_2_P4 	4
#define FUSION_SIZE_SLICE_2_H7 	16

#define FUSION_SIZE_INT_UNIT 	FUSION_SIZE_SLICE_1_H7

#define FUSION_SIZE_TB_1_X 		FUSION_SIZE_SLICE_1_H3 * FUSION_SIZE_SLICE_1_H2
#define FUSION_SIZE_TB_1_Y 		FUSION_SIZE_SLICE_1_P6 * FUSION_SIZE_SLICE_1_H1
#define FUSION_SIZE_REG_1_X 	FUSION_SIZE_SLICE_1_P5
#define FUSION_SIZE_REG_1_Y 	FUSION_SIZE_SLICE_1_P4

#define FUSION_SIZE_TB_2_X 		FUSION_SIZE_SLICE_2_H3 * FUSION_SIZE_SLICE_2_H2
#define FUSION_SIZE_TB_2_Y 		FUSION_SIZE_SLICE_2_P4 * FUSION_SIZE_SLICE_2_H1
#define FUSION_SIZE_REG_2_X 	FUSION_SIZE_SLICE_2_P5
#define FUSION_SIZE_REG_2_Y 	FUSION_SIZE_SLICE_2_P6

#define NUM_INDEX 	    		6
#define CEIL(a, b)      		(((a) + (b) - 1) / (b))

#define NUM_IA6_LOOPS           9
#define NUM_D1_EQUATIONS        9
#define NUM_D2_EQUATIONS        9
#define NUM_S1_EQUATIONS        9
#define NUM_D1_INDEX            7
#define NUM_D2_INDEX            7
#define NUM_S1_INDEX            6
#define NUM_ENERGIES            2
#define FULL_MASK 				0xffffffff

#define MAX_NOAB				10
#define MAX_NVAB 				30

// #define DEBUG_TIME_FUSED_CCSD_T
// #define DEBUG_KERNEL_DETAIL
// #define DEBUG_HOST_ENERGIES

// 64 KB = 65536 bytes = 16384 (int) = 8192 (size_t)
// 9 * 9 * noab = 81 * noab 
__constant__ int const_list_s1_flags_offset[NUM_IA6_LOOPS * NUM_S1_EQUATIONS];
__constant__ int const_list_d1_flags_offset[NUM_IA6_LOOPS * NUM_D1_EQUATIONS * MAX_NOAB];
__constant__ int const_list_d2_flags_offset[NUM_IA6_LOOPS * NUM_D2_EQUATIONS * MAX_NVAB];
// 
// __constant__ int const_s1_ia6[NUM_S1_EQUATIONS];			// 9 
// __constant__ int const_d1_ia6[NUM_D1_EQUATIONS * MAX_NOAB];	// 9 * noab
// __constant__ int const_d2_ia6[NUM_D2_EQUATIONS * MAX_NVAB];	// 9 * nvab
// // 
// __constant__ int const_s1_sizes_ext[NUM_IA6_LOOPS * NUM_S1_INDEX];	// 9 * 6
// __constant__ int const_d1_sizes_ext[NUM_IA6_LOOPS * NUM_D1_INDEX];	// 9 * 6
// __constant__ int const_d2_sizes_ext[NUM_IA6_LOOPS * NUM_D2_INDEX];	// 9 * 6
// // 
// __constant__ int const_s1_flags_offset[NUM_S1_EQUATIONS];
// __constant__ int const_d1_flags_offset[NUM_D1_EQUATIONS * MAX_NOAB];
// __constant__ int const_d2_flags_offset[NUM_D2_EQUATIONS * MAX_NVAB];
// 
__constant__ int const_list_s1_problem_size[NUM_IA6_LOOPS * NUM_S1_INDEX];
__constant__ int const_list_d1_problem_size[NUM_IA6_LOOPS * NUM_D1_INDEX * MAX_NOAB];
__constant__ int const_list_d2_problem_size[NUM_IA6_LOOPS * NUM_D2_INDEX * MAX_NVAB];

#if 0
	//  s1
	size_t size_s1_t2_1 = size_p4 * size_h1;
	size_t size_s1_v2_1 = size_h3 * size_h2 * size_p6 * size_p5;
	size_t size_s1_t2_2 = size_p4 * size_h2;
	size_t size_s1_v2_2 = size_h3 * size_h1 * size_p6 * size_p5;
	size_t size_s1_t2_3 = size_p4 * size_h1;
	size_t size_s1_v2_3 = size_h3 * size_h2 * size_p6 * size_p5;
	size_t size_s1_t2_4 = size_p5 * size_h1;
	size_t size_s1_v2_4 = size_h3 * size_h2 * size_p6 * size_p4;
	size_t size_s1_t2_5 = size_p5 * size_h2;
	size_t size_s1_v2_5 = size_h3 * size_h1 * size_p6 * size_p4;
	size_t size_s1_t2_6 = size_p5 * size_h3;
	size_t size_s1_v2_6 = size_h2 * size_h1 * size_p6 * size_p4;
	size_t size_s1_t2_7 = size_p6 * size_h1;
	size_t size_s1_v2_7 = size_h3 * size_h2 * size_p5 * size_p4;
	size_t size_s1_t2_8 = size_p6 * size_h2;
	size_t size_s1_v2_8 = size_h3 * size_h1 * size_p5 * size_p4;
	size_t size_s1_t2_9 = size_p6 * size_h3;
	size_t size_s1_v2_9 = size_h2 * size_h1 * size_p5 * size_p4;

	//  d1
	size_t size_d1_t2_1 = size_h1 * size_p5 * size_p4 * size_h7;
	size_t size_d1_v2_1 = size_h7 * size_p6 * size_h2 * size_h3;
	size_t size_d1_t2_2 = size_h2 * size_p5 * size_p4 * size_h7;
	size_t size_d1_v2_2 = size_h7 * size_p6 * size_h1 * size_h3;
	size_t size_d1_t2_3 = size_h3 * size_p5 * size_p4 * size_h7;
	size_t size_d1_v2_3 = size_h7 * size_p6 * size_h1 * size_h2;
	size_t size_d1_t2_4 = size_h1 * size_p6 * size_p5 * size_h7;
	size_t size_d1_v2_4 = size_h7 * size_p4 * size_h2 * size_h3;
	size_t size_d1_t2_5 = size_h2 * size_p6 * size_p5 * size_h7;
	size_t size_d1_v2_5 = size_h7 * size_p4 * size_h1 * size_h3;
	size_t size_d1_t2_6 = size_h3 * size_p6 * size_p5 * size_h7;
	size_t size_d1_v2_6 = size_h7 * size_p4 * size_h1 * size_h2;
	size_t size_d1_t2_7 = size_h1 * size_p6 * size_p4 * size_h7;
	size_t size_d1_v2_7 = size_h7 * size_p5 * size_h2 * size_h3;
	size_t size_d1_t2_8 = size_h2 * size_p6 * size_p4 * size_h7;
	size_t size_d1_v2_8 = size_h7 * size_p5 * size_h1 * size_h3;
	size_t size_d1_t2_9 = size_h3 * size_p6 * size_p4 * size_h7;
	size_t size_d1_v2_9 = size_h7 * size_p5 * size_h1 * size_h2;

	//  d2
	size_t size_d2_t2_1 = size_h2 * size_h1 * size_p4 * size_p7;
	size_t size_d2_v2_1 = size_p5 * size_p6 * size_h3 * size_p7;
	size_t size_d2_t2_2 = size_h3 * size_h2 * size_p4 * size_p7;
	size_t size_d2_v2_2 = size_p5 * size_p6 * size_h1 * size_p7;
	size_t size_d2_t2_3 = size_h3 * size_h1 * size_p4 * size_p7;
	size_t size_d2_v2_3 = size_p5 * size_p6 * size_h2 * size_p7;
	size_t size_d2_t2_4 = size_h2 * size_h1 * size_p5 * size_p7;
	size_t size_d2_v2_4 = size_p4 * size_p6 * size_h3 * size_p7;
	size_t size_d2_t2_5 = size_h3 * size_h2 * size_p5 * size_p7;
	size_t size_d2_v2_5 = size_p4 * size_p6 * size_h1 * size_p7;
	size_t size_d2_t2_6 = size_h3 * size_h1 * size_p5 * size_p7;
	size_t size_d2_v2_6 = size_p4 * size_p6 * size_h2 * size_p7;
	size_t size_d2_t2_7 = size_h2 * size_h1 * size_p6 * size_p7;
	size_t size_d2_v2_7 = size_p4 * size_p5 * size_h3 * size_p7;
	size_t size_d2_t2_8 = size_h3 * size_h2 * size_p6 * size_p7;
	size_t size_d2_v2_8 = size_p4 * size_p5 * size_h1 * size_p7;
	size_t size_d2_t2_9 = size_h3 * size_h1 * size_p6 * size_p7;
	size_t size_d2_v2_9 = size_p4 * size_p5 * size_h2 * size_p7;
#endif

// 
__global__ 
void jk_ccsd_t_fully_fused_kernel(	int size_noab, int size_nvab, 
									// 	common
									int size_max_dim_s1_t2, int size_max_dim_s1_v2, 
									int size_max_dim_d1_t2, int size_max_dim_d1_v2, 
									int size_max_dim_d2_t2, int size_max_dim_d2_v2, 
									//  doubles (sd1)
									double* dev_d1_t2_all,  double* dev_d1_v2_all,
									//  doubles (sd2)
									double* dev_d2_t2_all, double* dev_d2_v2_all,
									//  single 	(s1)
									double* dev_s1_t2_all, double* dev_s1_v2_all,
									//  energies
									const double* dev_evl_sorted_h1b, const double* dev_evl_sorted_h2b, const double* dev_evl_sorted_h3b,
									const double* dev_evl_sorted_p4b, const double* dev_evl_sorted_p5b, const double* dev_evl_sorted_p6b, 
									// 	not-fully reduced results
									double* reduced_energy,
									//  common
									int num_blks_h3b, int num_blks_h2b, int num_blks_h1b, 
									int num_blks_p6b, int num_blks_p5b, int num_blks_p4b, 
									int base_size_h1b, int base_size_h2b, int base_size_h3b, 
									int base_size_p4b, int base_size_p5b, int base_size_p6b)
{
	// For Shared Memory,
	__shared__ double sm_a[16][64 + 1];
	__shared__ double sm_b[16][64 + 1];
	
	int internal_upperbound = 0;
	int internal_offset;

	// should support for non-full tiles
	int idx_h3 			= threadIdx.x % FUSION_SIZE_SLICE_1_H3;
	int idx_h2 			= threadIdx.x / FUSION_SIZE_SLICE_1_H3;
	int idx_p6 			= threadIdx.y % FUSION_SIZE_SLICE_1_P6;
	int idx_h1 			= threadIdx.y / FUSION_SIZE_SLICE_1_P6;
	   
	int blk_idx_p4b     = blockIdx.x / (num_blks_h3b * num_blks_h2b * num_blks_h1b * num_blks_p6b * num_blks_p5b);
	int tmp_blkIdx      = blockIdx.x % (num_blks_h3b * num_blks_h2b * num_blks_h1b * num_blks_p6b * num_blks_p5b);
	int blk_idx_p5b     = (tmp_blkIdx) / (num_blks_h3b * num_blks_h2b * num_blks_h1b * num_blks_p6b);
	tmp_blkIdx          = (tmp_blkIdx) % (num_blks_h3b * num_blks_h2b * num_blks_h1b * num_blks_p6b);
	int blk_idx_p6b     = (tmp_blkIdx) / (num_blks_h3b * num_blks_h2b * num_blks_h1b);
	tmp_blkIdx          = (tmp_blkIdx) % (num_blks_h3b * num_blks_h2b * num_blks_h1b);
	int blk_idx_h1b     = (tmp_blkIdx) / (num_blks_h3b * num_blks_h2b);
	tmp_blkIdx          = (tmp_blkIdx) % (num_blks_h3b * num_blks_h2b);
	int blk_idx_h2b     = (tmp_blkIdx) / (num_blks_h3b);
	int blk_idx_h3b     = blockIdx.x % (num_blks_h3b);

	int str_blk_idx_h3 	= blk_idx_h3b * FUSION_SIZE_SLICE_1_H3;
	int str_blk_idx_h2 	= blk_idx_h2b * FUSION_SIZE_SLICE_1_H2;
	int str_blk_idx_h1 	= blk_idx_h1b * FUSION_SIZE_SLICE_1_H1;
	int str_blk_idx_p6 	= blk_idx_p6b * FUSION_SIZE_SLICE_1_P6;
	int str_blk_idx_p5 	= blk_idx_p5b * FUSION_SIZE_SLICE_1_P5;
	int str_blk_idx_p4 	= blk_idx_p4b * FUSION_SIZE_SLICE_1_P4;

	// 
	int rng_h3, rng_h2, rng_h1, rng_p6, rng_p5, rng_p4;
	int energy_rng_h3, energy_rng_h2, energy_rng_h1, energy_rng_p6, energy_rng_p5, energy_rng_p4;
	if ((base_size_h3b - (str_blk_idx_h3)) >= FUSION_SIZE_SLICE_1_H3)
	{
		energy_rng_h3 = FUSION_SIZE_SLICE_1_H3;
	}
	else
	{
		energy_rng_h3 = base_size_h3b % FUSION_SIZE_SLICE_1_H3;
	}
	
	if ((base_size_h2b - (str_blk_idx_h2)) >= FUSION_SIZE_SLICE_1_H2)
	{
		energy_rng_h2 = FUSION_SIZE_SLICE_1_H2;
	}
	else
	{
		energy_rng_h2 = base_size_h2b % FUSION_SIZE_SLICE_1_H2;
	}

	if ((base_size_h1b - (str_blk_idx_h1)) >= FUSION_SIZE_SLICE_1_H1)
	{
		energy_rng_h1 = FUSION_SIZE_SLICE_1_H1;
	}
	else
	{
		energy_rng_h1 = base_size_h1b % FUSION_SIZE_SLICE_1_H1;
	}
	
	if ((base_size_p6b - (str_blk_idx_p6)) >= FUSION_SIZE_SLICE_1_P6)
	{
		energy_rng_p6 = FUSION_SIZE_SLICE_1_P6;
	}
	else
	{
		energy_rng_p6 = base_size_p6b % FUSION_SIZE_SLICE_1_P6;
	}

	if ((base_size_p5b - (str_blk_idx_p5)) >= FUSION_SIZE_SLICE_1_P5)
	{
		energy_rng_p5 = FUSION_SIZE_SLICE_1_P5;
	}
	else
	{
		energy_rng_p5 = base_size_p5b % FUSION_SIZE_SLICE_1_P5;
	}

	if ((base_size_p4b - (str_blk_idx_p4)) >= FUSION_SIZE_SLICE_1_P4)
	{
		energy_rng_p4 = FUSION_SIZE_SLICE_1_P4;
	}
	else
	{
		energy_rng_p4 = base_size_p4b % FUSION_SIZE_SLICE_1_P4;
	}

	// 
	double temp_av;
	double temp_bv[4];
	double reg_tile[4][4];
	double reg_singles[4][4];

	int base_size_h7b, base_size_p7b;

	for (int i = 0; i < 4; i++)
	for (int j = 0; j < 4; j++)
	{
		reg_tile[i][j]      = 0.0;
		reg_singles[i][j]   = 0.0;
	}

	int energy_str_blk_idx_p4 = str_blk_idx_p4;
	int energy_str_blk_idx_p5 = str_blk_idx_p5;
	double eval_h3 = dev_evl_sorted_h3b[str_blk_idx_h3 + idx_h3];
	double eval_h2 = dev_evl_sorted_h2b[str_blk_idx_h2 + idx_h2];
	double eval_p6 = dev_evl_sorted_p6b[str_blk_idx_p6 + idx_p6];
	double eval_h1 = dev_evl_sorted_h1b[str_blk_idx_h1 + idx_h1];

	double partial_inner_factor = eval_h3 + eval_h2 + eval_h1 - eval_p6;

	// 
	//  loops
	// 
	#pragma unroll 1
	for (int iter_ia6 = 0; iter_ia6 < NUM_IA6_LOOPS; iter_ia6++)
	{
		//  doubles (d1 and d2) 
		{
			//  d1-top: sd1_1, 2 and 3 
			#pragma unroll 1
			for (int iter_noab = 0; iter_noab < size_noab; iter_noab++)
			{
				// 
				int flag_d1_1 = const_list_d1_flags_offset[0 + (iter_noab + (iter_ia6) * size_noab) * NUM_D1_EQUATIONS];
				int flag_d1_2 = const_list_d1_flags_offset[1 + (iter_noab + (iter_ia6) * size_noab) * NUM_D1_EQUATIONS];
				int flag_d1_3 = const_list_d1_flags_offset[2 + (iter_noab + (iter_ia6) * size_noab) * NUM_D1_EQUATIONS];

				// 
				// int local_d1_size_idx_h1b = const_list_d1_problem_size[0 + (iter_noab + (iter_ia6) * size_noab) * NUM_D1_INDEX];
				// int local_d1_size_idx_h2b = const_list_d1_problem_size[1 + (iter_noab + (iter_ia6) * size_noab) * NUM_D1_INDEX];
				// int local_d1_size_idx_h3b = const_list_d1_problem_size[2 + (iter_noab + (iter_ia6) * size_noab) * NUM_D1_INDEX];
				// // int local_d1_size_idx_h7b = const_list_d1_problem_size[3 + (iter_noab + (iter_ia6) * size_noab) * NUM_D1_INDEX];
				// int local_d1_size_idx_p4b = const_list_d1_problem_size[4 + (iter_noab + (iter_ia6) * size_noab) * NUM_D1_INDEX];
				// int local_d1_size_idx_p5b = const_list_d1_problem_size[5 + (iter_noab + (iter_ia6) * size_noab) * NUM_D1_INDEX];
				// int local_d1_size_idx_p6b = const_list_d1_problem_size[6 + (iter_noab + (iter_ia6) * size_noab) * NUM_D1_INDEX];
				base_size_h1b = const_list_d1_problem_size[0 + (iter_noab + (iter_ia6) * size_noab) * NUM_D1_INDEX];
				base_size_h2b = const_list_d1_problem_size[1 + (iter_noab + (iter_ia6) * size_noab) * NUM_D1_INDEX];
				base_size_h3b = const_list_d1_problem_size[2 + (iter_noab + (iter_ia6) * size_noab) * NUM_D1_INDEX];
				base_size_h7b = const_list_d1_problem_size[3 + (iter_noab + (iter_ia6) * size_noab) * NUM_D1_INDEX];
				base_size_p4b = const_list_d1_problem_size[4 + (iter_noab + (iter_ia6) * size_noab) * NUM_D1_INDEX];
				base_size_p5b = const_list_d1_problem_size[5 + (iter_noab + (iter_ia6) * size_noab) * NUM_D1_INDEX];
				base_size_p6b = const_list_d1_problem_size[6 + (iter_noab + (iter_ia6) * size_noab) * NUM_D1_INDEX];

				// 
				// if (blockIdx.x == 0 && threadIdx.x == 0 && threadIdx.y == 0)
				// {
				// 	printf ("[sd1][part1][ia6 = %2d] h1,h2,h3,p4,p5,p6 = %2d,%2d,%2d,%2d,%2d,%2d\n", iter_ia6, base_size_h1b, base_size_h2b, base_size_h3b, base_size_p4b, base_size_p5b, base_size_p6b);
				// }

				//	otheres according to the above problem-sizes
				//	(1) num_blks_h/p*b
				// num_blks_h1b = CEIL(local_d1_size_idx_h1b, FUSION_SIZE_SLICE_1_H1);
				// num_blks_h2b = CEIL(local_d1_size_idx_h2b, FUSION_SIZE_SLICE_1_H2);
				// num_blks_h3b = CEIL(local_d1_size_idx_h3b, FUSION_SIZE_SLICE_1_H3);
				// num_blks_p4b = CEIL(local_d1_size_idx_p4b, FUSION_SIZE_SLICE_1_P4);
				// num_blks_p5b = CEIL(local_d1_size_idx_p5b, FUSION_SIZE_SLICE_1_P5);
				// num_blks_p6b = CEIL(local_d1_size_idx_p6b, FUSION_SIZE_SLICE_1_P6);
				num_blks_h1b = CEIL(base_size_h1b, FUSION_SIZE_SLICE_1_H1);
				num_blks_h2b = CEIL(base_size_h2b, FUSION_SIZE_SLICE_1_H2);
				num_blks_h3b = CEIL(base_size_h3b, FUSION_SIZE_SLICE_1_H3);
				num_blks_p4b = CEIL(base_size_p4b, FUSION_SIZE_SLICE_1_P4);
				num_blks_p5b = CEIL(base_size_p5b, FUSION_SIZE_SLICE_1_P5);
				num_blks_p6b = CEIL(base_size_p6b, FUSION_SIZE_SLICE_1_P6);

				// 	(2) blk_idx_h/p*b
				blk_idx_p4b     = blockIdx.x / (num_blks_h3b * num_blks_h2b * num_blks_h1b * num_blks_p6b * num_blks_p5b);
				tmp_blkIdx      = blockIdx.x % (num_blks_h3b * num_blks_h2b * num_blks_h1b * num_blks_p6b * num_blks_p5b);
				blk_idx_p5b     = (tmp_blkIdx) / (num_blks_h3b * num_blks_h2b * num_blks_h1b * num_blks_p6b);
				tmp_blkIdx  	= (tmp_blkIdx) % (num_blks_h3b * num_blks_h2b * num_blks_h1b * num_blks_p6b);
				blk_idx_p6b     = (tmp_blkIdx) / (num_blks_h3b * num_blks_h2b * num_blks_h1b);
				tmp_blkIdx 		= (tmp_blkIdx) % (num_blks_h3b * num_blks_h2b * num_blks_h1b);
				blk_idx_h1b     = (tmp_blkIdx) / (num_blks_h3b * num_blks_h2b);
				tmp_blkIdx 		= (tmp_blkIdx) % (num_blks_h3b * num_blks_h2b);
				blk_idx_h2b 	= (tmp_blkIdx) / (num_blks_h3b);
				blk_idx_h3b		= blockIdx.x % (num_blks_h3b);

				// 	(3) str_blk_idx_h/p*
				str_blk_idx_h3 	= blk_idx_h3b * FUSION_SIZE_SLICE_1_H3;
				str_blk_idx_h2 	= blk_idx_h2b * FUSION_SIZE_SLICE_1_H2;
				str_blk_idx_h1 	= blk_idx_h1b * FUSION_SIZE_SLICE_1_H1;
				str_blk_idx_p6 	= blk_idx_p6b * FUSION_SIZE_SLICE_1_P6;
				str_blk_idx_p5 	= blk_idx_p5b * FUSION_SIZE_SLICE_1_P5;
				str_blk_idx_p4 	= blk_idx_p4b * FUSION_SIZE_SLICE_1_P4;

				// 	(4) rng_h/p*
				if ((base_size_h3b - (str_blk_idx_h3)) >= FUSION_SIZE_SLICE_1_H3)
					rng_h3 = FUSION_SIZE_SLICE_1_H3;
				else
					rng_h3 = base_size_h3b % FUSION_SIZE_SLICE_1_H3;
				
				if ((base_size_h2b - (str_blk_idx_h2)) >= FUSION_SIZE_SLICE_1_H2)
					rng_h2 = FUSION_SIZE_SLICE_1_H2;
				else
					rng_h2 = base_size_h2b % FUSION_SIZE_SLICE_1_H2;

				if ((base_size_h1b - (str_blk_idx_h1)) >= FUSION_SIZE_SLICE_1_H1)
					rng_h1 = FUSION_SIZE_SLICE_1_H1;
				else
					rng_h1 = base_size_h1b % FUSION_SIZE_SLICE_1_H1;
				
				if ((base_size_p6b - (str_blk_idx_p6)) >= FUSION_SIZE_SLICE_1_P6)
					rng_p6 = FUSION_SIZE_SLICE_1_P6;
				else
					rng_p6 = base_size_p6b % FUSION_SIZE_SLICE_1_P6;

				if ((base_size_p5b - (str_blk_idx_p5)) >= FUSION_SIZE_SLICE_1_P5)
					rng_p5 = FUSION_SIZE_SLICE_1_P5;
				else
					rng_p5 = base_size_p5b % FUSION_SIZE_SLICE_1_P5;

				if ((base_size_p4b - (str_blk_idx_p4)) >= FUSION_SIZE_SLICE_1_P4)
					rng_p4 = FUSION_SIZE_SLICE_1_P4;
				else
					rng_p4 = base_size_p4b % FUSION_SIZE_SLICE_1_P4;
			
				//  sd1_1
				if (flag_d1_1 >= 0)
				{
					// 
					double* tmp_dev_d1_t2 = dev_d1_t2_all + size_max_dim_d1_t2 * flag_d1_1;
					double* tmp_dev_d1_v2 = dev_d1_v2_all + size_max_dim_d1_v2 * flag_d1_1;

					// 
					#pragma unroll 1
					for (int l = 0; l < base_size_h7b; l+= FUSION_SIZE_INT_UNIT)
					{
						// Part: Generalized Contraction Index (p7b)
						internal_offset = (l + FUSION_SIZE_INT_UNIT) - base_size_h7b;
						if (internal_offset > 0) internal_upperbound = internal_offset;

						// Load Input Tensor to Shared Memory: 16:16
						// # of size_internal Indices: 1
						if (idx_p6 < rng_p4 && idx_h1 < rng_h1 && threadIdx.x < FUSION_SIZE_INT_UNIT - internal_upperbound)
						for (int ll = 0; ll < rng_p5; ll++)
						{
							sm_a[threadIdx.x][threadIdx.y + ll * FUSION_SIZE_TB_2_Y] = tmp_dev_d1_t2[(str_blk_idx_p4 + idx_p6 + (str_blk_idx_p5 + ll + (str_blk_idx_h1 + idx_h1) * base_size_p5b) * base_size_p4b) * base_size_h7b + (threadIdx.x + l)];
						}

						// Load Input Tensor to Shared Memory
						if (idx_h3 < rng_h3 && idx_h2 < rng_h2 && threadIdx.y < FUSION_SIZE_INT_UNIT - internal_upperbound)
						for (int ll = 0; ll < rng_p6; ll++)
						{
							sm_b[threadIdx.y][threadIdx.x + ll * FUSION_SIZE_TB_2_X] = tmp_dev_d1_v2[str_blk_idx_h3 + idx_h3 + (str_blk_idx_h2 + idx_h2 + (str_blk_idx_p6 + ll + (threadIdx.y + l) * base_size_p6b) * base_size_h2b) * base_size_h3b];
						}
						__syncthreads();

						// Cross-Product: -1
						// Part: Generalized Threads
						for (int ll = 0; ll < FUSION_SIZE_INT_UNIT - internal_upperbound; ll++)
						{
							temp_bv[0] = sm_b[ll][idx_h3 + (idx_h2) * FUSION_SIZE_SLICE_2_H3 + 0];
							temp_bv[1] = sm_b[ll][idx_h3 + (idx_h2) * FUSION_SIZE_SLICE_2_H3 + 16];
							temp_bv[2] = sm_b[ll][idx_h3 + (idx_h2) * FUSION_SIZE_SLICE_2_H3 + 32];
							temp_bv[3] = sm_b[ll][idx_h3 + (idx_h2) * FUSION_SIZE_SLICE_2_H3 + 48];

							for (int xx = 0 ; xx < 4; xx++)
							{
								temp_av = sm_a[ll][idx_p6 + (idx_h1) * FUSION_SIZE_SLICE_1_P6 + (xx * 16)];

								reg_tile[0][xx] -= temp_av * temp_bv[0];
								reg_tile[1][xx] -= temp_av * temp_bv[1];
								reg_tile[2][xx] -= temp_av * temp_bv[2];
								reg_tile[3][xx] -= temp_av * temp_bv[3];
							}
						}
						__syncthreads();
					}
				}
			
				//  sd1_2
				if (flag_d1_2 >= 0)
				{
					// 
					double* tmp_dev_d1_t2 = dev_d1_t2_all + size_max_dim_d1_t2 * flag_d1_2;
					double* tmp_dev_d1_v2 = dev_d1_v2_all + size_max_dim_d1_v2 * flag_d1_2;

					internal_upperbound = 0;
					#pragma unroll 1
					for (int l = 0; l < base_size_h7b; l+= FUSION_SIZE_INT_UNIT)
					{
						// Part: Generalized Contraction Index (p7b)
						internal_offset = (l + FUSION_SIZE_INT_UNIT) - base_size_h7b;
						if (internal_offset > 0) internal_upperbound = internal_offset;

						// Load Input Tensor to Shared Memory: 16:16
						// # of size_internal Indices: 1
						if (idx_p6 < rng_p4 && idx_h1 < rng_h2 && threadIdx.x < FUSION_SIZE_INT_UNIT - internal_upperbound)
						for (int ll = 0; ll < rng_p5; ll++)
						{
							sm_a[threadIdx.x][threadIdx.y + ll * FUSION_SIZE_TB_2_Y] = tmp_dev_d1_t2[(str_blk_idx_p4 + idx_p6 + (str_blk_idx_p5 + ll + (str_blk_idx_h2 + idx_h1) * base_size_p5b) * base_size_p4b) * base_size_h7b + (threadIdx.x + l)];
						}

						// Load Input Tensor to Shared Memory
						if (idx_h3 < rng_h3 && idx_h2 < rng_h1 && threadIdx.y < FUSION_SIZE_INT_UNIT - internal_upperbound)
						for (int ll = 0; ll < rng_p6; ll++)
						{
							sm_b[threadIdx.y][threadIdx.x + ll * FUSION_SIZE_TB_2_X] = tmp_dev_d1_v2[str_blk_idx_h3 + idx_h3 + (str_blk_idx_h1 + idx_h2 + (str_blk_idx_p6 + ll + (threadIdx.y + l) * base_size_p6b) * base_size_h1b) * base_size_h3b]; 
						}
						__syncthreads();

						// Cross-Product: -1
						// Part: Generalized Threads
						for (int ll = 0; ll < FUSION_SIZE_INT_UNIT - internal_upperbound; ll++)
						{
							temp_bv[0] = sm_b[ll][idx_h3 + (idx_h1) * FUSION_SIZE_SLICE_2_H3 + 0];
							temp_bv[1] = sm_b[ll][idx_h3 + (idx_h1) * FUSION_SIZE_SLICE_2_H3 + 16];
							temp_bv[2] = sm_b[ll][idx_h3 + (idx_h1) * FUSION_SIZE_SLICE_2_H3 + 32];
							temp_bv[3] = sm_b[ll][idx_h3 + (idx_h1) * FUSION_SIZE_SLICE_2_H3 + 48];

							for (int xx = 0 ; xx < 4; xx++)
							{
								temp_av = sm_a[ll][idx_p6 + (idx_h2) * FUSION_SIZE_SLICE_2_P4 + (xx * 16)];

								reg_tile[0][xx] += temp_av * temp_bv[0];
								reg_tile[1][xx] += temp_av * temp_bv[1];
								reg_tile[2][xx] += temp_av * temp_bv[2];
								reg_tile[3][xx] += temp_av * temp_bv[3];
							}
						}
						__syncthreads();
					}
				}
			
				//  sd1_3
				if (flag_d1_3 >= 0)
				{
					// 
					double* tmp_dev_d1_t2 = dev_d1_t2_all + size_max_dim_d1_t2 * flag_d1_3;
					double* tmp_dev_d1_v2 = dev_d1_v2_all + size_max_dim_d1_v2 * flag_d1_3;
					
					internal_upperbound = 0;
					#pragma unroll 1
					for (int l = 0; l < base_size_h7b; l+= FUSION_SIZE_INT_UNIT)
					{
						// Part: Generalized Contraction Index (p7b)
						internal_offset = (l + FUSION_SIZE_INT_UNIT) - base_size_h7b;
						if (internal_offset > 0) internal_upperbound = internal_offset;

						// Load Input Tensor to Shared Memory: 16:16
						// # of size_internal Indices: 1
						if (idx_p6 < rng_p4 && idx_h1 < rng_h3 && threadIdx.x < FUSION_SIZE_INT_UNIT - internal_upperbound)
						for (int ll = 0; ll < rng_p5; ll++)
						{
							sm_a[threadIdx.x][threadIdx.y + ll * FUSION_SIZE_TB_2_Y] = tmp_dev_d1_t2[(str_blk_idx_p4 + idx_p6 + (str_blk_idx_p5 + ll + (str_blk_idx_h3 + idx_h1) * base_size_p5b) * base_size_p4b) * base_size_h7b + (threadIdx.x + l)];
						}

						// Load Input Tensor to Shared Memory
						if (idx_h3 < rng_h2 && idx_h2 < rng_h1 && threadIdx.y < FUSION_SIZE_INT_UNIT - internal_upperbound)
						for (int ll = 0; ll < rng_p6; ll++)
						{
							sm_b[threadIdx.y][threadIdx.x + ll * FUSION_SIZE_TB_2_X] = tmp_dev_d1_v2[(str_blk_idx_h2 + idx_h3 + (str_blk_idx_h1 + idx_h2 + (str_blk_idx_p6 + ll + (threadIdx.y + l) * base_size_p6b) * base_size_h1b) * base_size_h2b)];
						}
						__syncthreads();

						// Cross-Product: -1
						// Part: Generalized Threads
						for (int ll = 0; ll < FUSION_SIZE_INT_UNIT - internal_upperbound; ll++)
						{
							temp_bv[0] = sm_b[ll][idx_h2 + (idx_h1) * FUSION_SIZE_SLICE_2_H2 + 0];
							temp_bv[1] = sm_b[ll][idx_h2 + (idx_h1) * FUSION_SIZE_SLICE_2_H2 + 16];
							temp_bv[2] = sm_b[ll][idx_h2 + (idx_h1) * FUSION_SIZE_SLICE_2_H2 + 32];
							temp_bv[3] = sm_b[ll][idx_h2 + (idx_h1) * FUSION_SIZE_SLICE_2_H2 + 48];

							for (int xx = 0 ; xx < 4; xx++)
							{
								temp_av = sm_a[ll][idx_p6 + (idx_h3) * FUSION_SIZE_SLICE_2_P4 + (xx * 16)];

								reg_tile[0][xx] -= temp_av * temp_bv[0];
								reg_tile[1][xx] -= temp_av * temp_bv[1];
								reg_tile[2][xx] -= temp_av * temp_bv[2];
								reg_tile[3][xx] -= temp_av * temp_bv[3];
							}
						}
						__syncthreads();
					}
				}
			}
		
			//  d2-top: sd2_7, 8 and 9
			#pragma unroll 1
			for (int iter_nvab = 0; iter_nvab < size_nvab; iter_nvab++)
			{
				// 
				int flag_d2_7 = const_list_d2_flags_offset[6 + (iter_nvab + (iter_ia6) * size_nvab) * NUM_D2_EQUATIONS];
				int flag_d2_8 = const_list_d2_flags_offset[7 + (iter_nvab + (iter_ia6) * size_nvab) * NUM_D2_EQUATIONS];
				int flag_d2_9 = const_list_d2_flags_offset[8 + (iter_nvab + (iter_ia6) * size_nvab) * NUM_D2_EQUATIONS];

				// 
				// int local_d2_size_idx_h1b = const_list_d2_problem_size[0 + (iter_nvab + (iter_ia6) * size_nvab) * NUM_D2_INDEX];
				// int local_d2_size_idx_h2b = const_list_d2_problem_size[1 + (iter_nvab + (iter_ia6) * size_nvab) * NUM_D2_INDEX];
				// int local_d2_size_idx_h3b = const_list_d2_problem_size[2 + (iter_nvab + (iter_ia6) * size_nvab) * NUM_D2_INDEX];
				// int local_d2_size_idx_p4b = const_list_d2_problem_size[3 + (iter_nvab + (iter_ia6) * size_nvab) * NUM_D2_INDEX];
				// int local_d2_size_idx_p5b = const_list_d2_problem_size[4 + (iter_nvab + (iter_ia6) * size_nvab) * NUM_D2_INDEX];
				// int local_d2_size_idx_p6b = const_list_d2_problem_size[5 + (iter_nvab + (iter_ia6) * size_nvab) * NUM_D2_INDEX];
				// int local_d2_size_idx_p7b = const_list_d2_problem_size[6 + (iter_nvab + (iter_ia6) * size_nvab) * NUM_D2_INDEX];
				base_size_h1b = const_list_d2_problem_size[0 + (iter_nvab + (iter_ia6) * size_nvab) * NUM_D2_INDEX];
				base_size_h2b = const_list_d2_problem_size[1 + (iter_nvab + (iter_ia6) * size_nvab) * NUM_D2_INDEX];
				base_size_h3b = const_list_d2_problem_size[2 + (iter_nvab + (iter_ia6) * size_nvab) * NUM_D2_INDEX];
				base_size_p4b = const_list_d2_problem_size[3 + (iter_nvab + (iter_ia6) * size_nvab) * NUM_D2_INDEX];
				base_size_p5b = const_list_d2_problem_size[4 + (iter_nvab + (iter_ia6) * size_nvab) * NUM_D2_INDEX];
				base_size_p6b = const_list_d2_problem_size[5 + (iter_nvab + (iter_ia6) * size_nvab) * NUM_D2_INDEX];
				base_size_p7b = const_list_d2_problem_size[6 + (iter_nvab + (iter_ia6) * size_nvab) * NUM_D2_INDEX];

				// // 
				// if (blockIdx.x == 0 && threadIdx.x == 0 && threadIdx.y == 0)
				// {
				// 	printf ("[sd2][part1][ia6 = %2d] h1,h2,h3,p4,p5,p6 = %2d,%2d,%2d,%2d,%2d,%2d\n", iter_ia6, base_size_h1b, base_size_h2b, base_size_h3b, base_size_p4b, base_size_p5b, base_size_p6b);
				// }

				//	otheres according to the above problem-sizes
				//	(1) num_blks_h/p*b
				// num_blks_h1b = CEIL(local_d2_size_idx_h1b, FUSION_SIZE_SLICE_1_H1);
				// num_blks_h2b = CEIL(local_d2_size_idx_h2b, FUSION_SIZE_SLICE_1_H2);
				// num_blks_h3b = CEIL(local_d2_size_idx_h3b, FUSION_SIZE_SLICE_1_H3);
				// num_blks_p4b = CEIL(local_d2_size_idx_p4b, FUSION_SIZE_SLICE_1_P4);
				// num_blks_p5b = CEIL(local_d2_size_idx_p5b, FUSION_SIZE_SLICE_1_P5);
				// num_blks_p6b = CEIL(local_d2_size_idx_p6b, FUSION_SIZE_SLICE_1_P6);
				num_blks_h1b = CEIL(base_size_h1b, FUSION_SIZE_SLICE_1_H1);
				num_blks_h2b = CEIL(base_size_h2b, FUSION_SIZE_SLICE_1_H2);
				num_blks_h3b = CEIL(base_size_h3b, FUSION_SIZE_SLICE_1_H3);
				num_blks_p4b = CEIL(base_size_p4b, FUSION_SIZE_SLICE_1_P4);
				num_blks_p5b = CEIL(base_size_p5b, FUSION_SIZE_SLICE_1_P5);
				num_blks_p6b = CEIL(base_size_p6b, FUSION_SIZE_SLICE_1_P6);

				// 	(2) blk_idx_h/p*b
				blk_idx_p4b     = blockIdx.x / (num_blks_h3b * num_blks_h2b * num_blks_h1b * num_blks_p6b * num_blks_p5b);
				tmp_blkIdx      = blockIdx.x % (num_blks_h3b * num_blks_h2b * num_blks_h1b * num_blks_p6b * num_blks_p5b);
				blk_idx_p5b     = (tmp_blkIdx) / (num_blks_h3b * num_blks_h2b * num_blks_h1b * num_blks_p6b);
				tmp_blkIdx  	= (tmp_blkIdx) % (num_blks_h3b * num_blks_h2b * num_blks_h1b * num_blks_p6b);
				blk_idx_p6b     = (tmp_blkIdx) / (num_blks_h3b * num_blks_h2b * num_blks_h1b);
				tmp_blkIdx 		= (tmp_blkIdx) % (num_blks_h3b * num_blks_h2b * num_blks_h1b);
				blk_idx_h1b     = (tmp_blkIdx) / (num_blks_h3b * num_blks_h2b);
				tmp_blkIdx 		= (tmp_blkIdx) % (num_blks_h3b * num_blks_h2b);
				blk_idx_h2b 	= (tmp_blkIdx) / (num_blks_h3b);
				blk_idx_h3b		= blockIdx.x % (num_blks_h3b);

				// 	(3) str_blk_idx_h/p*
				str_blk_idx_h3 	= blk_idx_h3b * FUSION_SIZE_SLICE_1_H3;
				str_blk_idx_h2 	= blk_idx_h2b * FUSION_SIZE_SLICE_1_H2;
				str_blk_idx_h1 	= blk_idx_h1b * FUSION_SIZE_SLICE_1_H1;
				str_blk_idx_p6 	= blk_idx_p6b * FUSION_SIZE_SLICE_1_P6;
				str_blk_idx_p5 	= blk_idx_p5b * FUSION_SIZE_SLICE_1_P5;
				str_blk_idx_p4 	= blk_idx_p4b * FUSION_SIZE_SLICE_1_P4;

				// 	(4) rng_h/p*
				if ((base_size_h3b - (str_blk_idx_h3)) >= FUSION_SIZE_SLICE_1_H3)
					rng_h3 = FUSION_SIZE_SLICE_1_H3;
				else
					rng_h3 = base_size_h3b % FUSION_SIZE_SLICE_1_H3;
				
				if ((base_size_h2b - (str_blk_idx_h2)) >= FUSION_SIZE_SLICE_1_H2)
					rng_h2 = FUSION_SIZE_SLICE_1_H2;
				else
					rng_h2 = base_size_h2b % FUSION_SIZE_SLICE_1_H2;

				if ((base_size_h1b - (str_blk_idx_h1)) >= FUSION_SIZE_SLICE_1_H1)
					rng_h1 = FUSION_SIZE_SLICE_1_H1;
				else
					rng_h1 = base_size_h1b % FUSION_SIZE_SLICE_1_H1;
				
				if ((base_size_p6b - (str_blk_idx_p6)) >= FUSION_SIZE_SLICE_1_P6)
					rng_p6 = FUSION_SIZE_SLICE_1_P6;
				else
					rng_p6 = base_size_p6b % FUSION_SIZE_SLICE_1_P6;

				if ((base_size_p5b - (str_blk_idx_p5)) >= FUSION_SIZE_SLICE_1_P5)
					rng_p5 = FUSION_SIZE_SLICE_1_P5;
				else
					rng_p5 = base_size_p5b % FUSION_SIZE_SLICE_1_P5;

				if ((base_size_p4b - (str_blk_idx_p4)) >= FUSION_SIZE_SLICE_1_P4)
					rng_p4 = FUSION_SIZE_SLICE_1_P4;
				else
					rng_p4 = base_size_p4b % FUSION_SIZE_SLICE_1_P4;

				//	sd2_7
				if (flag_d2_7 >= 0)
				{
					// 
					double* tmp_dev_d2_t2_7 = dev_d2_t2_all + size_max_dim_d2_t2 * flag_d2_7;//const_list_d2_flags_offset[local_offset];
					double* tmp_dev_d2_v2_7 = dev_d2_v2_all + size_max_dim_d2_v2 * flag_d2_7;//const_list_d2_flags_offset[local_offset];
					
					//	sd2_7
					internal_upperbound = 0;
					#pragma unroll 1
					for (int l = 0; l < base_size_p7b; l+= FUSION_SIZE_INT_UNIT)
					{
						// Part: Generalized Contraction Index (p7b)
						internal_offset = (l + FUSION_SIZE_INT_UNIT) - base_size_p7b;
						if (internal_offset > 0) internal_upperbound = internal_offset;

						// Load Input Tensor to Shared Memory: 16:16
						// # of size_internal Indices: 1
						if (idx_p6 < rng_h1 && idx_h1 < rng_h2 && threadIdx.x < FUSION_SIZE_INT_UNIT - internal_upperbound)
						for (int ll = 0; ll < rng_p6; ll++)
						{
							sm_a[threadIdx.x][threadIdx.y + ll * FUSION_SIZE_TB_2_Y] = tmp_dev_d2_t2_7[(blk_idx_p6b *  FUSION_SIZE_SLICE_2_P6 + ll + (str_blk_idx_h1 + idx_p6 + (str_blk_idx_h2 + idx_h1) * base_size_h1b) * base_size_p6b) * base_size_p7b + (threadIdx.x + l)];
						}

						// Load Input Tensor to Shared Memory
						if (idx_p6 < rng_h3 && idx_h1 < rng_p4 && threadIdx.x < FUSION_SIZE_INT_UNIT - internal_upperbound)
						for (int ll = 0; ll < rng_p5; ll++)
						{
							sm_b[threadIdx.x][threadIdx.y + ll * FUSION_SIZE_TB_2_Y] = tmp_dev_d2_v2_7[(str_blk_idx_h3 + idx_p6 + (str_blk_idx_p5 + ll + (str_blk_idx_p4 + idx_h1) * base_size_p5b) * base_size_h3b) * base_size_p7b + (threadIdx.x + l)];
						}
						__syncthreads();

						// Cross-Product: 16
						// Part: Generalized Threads
						for (int ll = 0; ll < FUSION_SIZE_INT_UNIT - internal_upperbound; ll++)
						{
							temp_bv[0] = sm_a[ll][idx_h1 + (idx_h2) * FUSION_SIZE_SLICE_2_H1 + 0];
							temp_bv[1] = sm_a[ll][idx_h1 + (idx_h2) * FUSION_SIZE_SLICE_2_H1 + 16];
							temp_bv[2] = sm_a[ll][idx_h1 + (idx_h2) * FUSION_SIZE_SLICE_2_H1 + 32];
							temp_bv[3] = sm_a[ll][idx_h1 + (idx_h2) * FUSION_SIZE_SLICE_2_H1 + 48];

							for (int xx = 0 ; xx < 4; xx++)
							{
								temp_av = sm_b[ll][idx_h3 + (idx_p6) * FUSION_SIZE_SLICE_2_H3 + (xx * 16)];

								reg_tile[0][xx] -= temp_av * temp_bv[0];
								reg_tile[1][xx] -= temp_av * temp_bv[1];
								reg_tile[2][xx] -= temp_av * temp_bv[2];
								reg_tile[3][xx] -= temp_av * temp_bv[3];
							}
						}
						__syncthreads();
					}
				}

				// 	sd2_8
				if (flag_d2_8 >= 0)
				{
					// 
					double* tmp_dev_d2_t2_8 = dev_d2_t2_all + size_max_dim_d2_t2 * flag_d2_8;//const_list_d2_flags_offset[local_offset];
					double* tmp_dev_d2_v2_8 = dev_d2_v2_all + size_max_dim_d2_v2 * flag_d2_8;//const_list_d2_flags_offset[local_offset];

					internal_upperbound = 0;
					#pragma unroll 1
					for (int l = 0; l < base_size_p7b; l+= FUSION_SIZE_INT_UNIT)
					{
						// Part: Generalized Contraction Index (p7b)
						// internal_offset = (l + FUSION_SIZE_INT_UNIT) - size_internal;
						internal_offset = (l + FUSION_SIZE_INT_UNIT) - base_size_p7b;
						if (internal_offset > 0) internal_upperbound = internal_offset;
				
						// Load Input Tensor to Shared Memory: 16:16
						// # of size_internal Indices: 1
						if (idx_p6 < rng_h2 && idx_h1 < rng_h3 && threadIdx.x < FUSION_SIZE_INT_UNIT - internal_upperbound)
						for (int ll = 0; ll < rng_p6; ll++)
						{
							sm_a[threadIdx.x][threadIdx.y + ll * FUSION_SIZE_TB_2_Y] = tmp_dev_d2_t2_8[(str_blk_idx_p6 + ll + (str_blk_idx_h2 + idx_p6 + (str_blk_idx_h3 + idx_h1) * base_size_h2b) * base_size_p6b) * base_size_p7b + (threadIdx.x + l)];
						}
				
						// Load Input Tensor to Shared Memory
						if (idx_p6 < rng_h1 && idx_h1 < rng_p4 && threadIdx.x < FUSION_SIZE_INT_UNIT - internal_upperbound)
						for (int ll = 0; ll < rng_p5; ll++)
						{
							sm_b[threadIdx.x][threadIdx.y + ll * FUSION_SIZE_TB_2_Y] = tmp_dev_d2_v2_8[(str_blk_idx_h1 + idx_p6 + (str_blk_idx_p5 + ll + (str_blk_idx_p4 + idx_h1) * base_size_p5b) * base_size_h1b) * base_size_p7b + (threadIdx.x + l)];
						}
						__syncthreads();
				
						// Cross-Product: 16
						// Part: Generalized Threads
						for (int ll = 0; ll < FUSION_SIZE_INT_UNIT - internal_upperbound; ll++)
						{
							temp_bv[0] = sm_a[ll][idx_h2 + (idx_h3) * FUSION_SIZE_SLICE_2_H2 + 0];
							temp_bv[1] = sm_a[ll][idx_h2 + (idx_h3) * FUSION_SIZE_SLICE_2_H2 + 16];
							temp_bv[2] = sm_a[ll][idx_h2 + (idx_h3) * FUSION_SIZE_SLICE_2_H2 + 32];
							temp_bv[3] = sm_a[ll][idx_h2 + (idx_h3) * FUSION_SIZE_SLICE_2_H2 + 48];
				
							for (int xx = 0 ; xx < 4; xx++)
							{
								temp_av = sm_b[ll][idx_h1 + (idx_p6) * FUSION_SIZE_SLICE_2_H1 + (xx * 16)];
				
								reg_tile[0][xx] -= temp_av * temp_bv[0];
								reg_tile[1][xx] -= temp_av * temp_bv[1];
								reg_tile[2][xx] -= temp_av * temp_bv[2];
								reg_tile[3][xx] -= temp_av * temp_bv[3];
							}
						}
						__syncthreads();
					}
				}
			
				// 	sd2_9
				if (flag_d2_9 >= 0)
				{
					// 
					double* tmp_dev_d2_t2_9 = dev_d2_t2_all + size_max_dim_d2_t2 * flag_d2_9;//const_list_d2_flags_offset[local_offset];
					double* tmp_dev_d2_v2_9 = dev_d2_v2_all + size_max_dim_d2_v2 * flag_d2_9;//const_list_d2_flags_offset[local_offset];

					internal_upperbound = 0;
					#pragma unroll 1
					for (int l = 0; l < base_size_p7b; l+= FUSION_SIZE_INT_UNIT)
					{
						// Part: Generalized Contraction Index (p7b)
						// internal_offset = (l + FUSION_SIZE_INT_UNIT) - size_internal;
						internal_offset = (l + FUSION_SIZE_INT_UNIT) - base_size_p7b;
						if (internal_offset > 0) internal_upperbound = internal_offset;
				
						// Load Input Tensor to Shared Memory: 16:16
						// # of size_internal Indices: 1
						if (idx_p6 < rng_h1 && idx_h1 < rng_h3 && threadIdx.x < FUSION_SIZE_INT_UNIT - internal_upperbound)
						for (int ll = 0; ll < rng_p6; ll++)
						{
							sm_a[threadIdx.x][threadIdx.y + ll * FUSION_SIZE_TB_2_Y] = tmp_dev_d2_t2_9[(str_blk_idx_p6 + ll + (str_blk_idx_h1 + idx_p6 + (str_blk_idx_h3 + idx_h1) * base_size_h1b) * base_size_p6b) * base_size_p7b + (threadIdx.x + l)];
						}
				
						// Load Input Tensor to Shared Memory
						if (idx_p6 < rng_h2 && idx_h1 < rng_p4 && threadIdx.x < FUSION_SIZE_INT_UNIT - internal_upperbound)
						for (int ll = 0; ll < rng_p5; ll++)
						{
							sm_b[threadIdx.x][threadIdx.y + ll * FUSION_SIZE_TB_2_Y] = tmp_dev_d2_v2_9[(str_blk_idx_h2 + idx_p6 + (str_blk_idx_p5 + ll + (str_blk_idx_p4 + idx_h1) * base_size_p5b) * base_size_h2b) * base_size_p7b + (threadIdx.x + l)];
						}
						__syncthreads();
				
						// Cross-Product: 16
						// Part: Generalized Threads
						for (int ll = 0; ll < FUSION_SIZE_INT_UNIT - internal_upperbound; ll++)
						{
							temp_bv[0] = sm_a[ll][idx_h1 + (idx_h3) * FUSION_SIZE_SLICE_2_H1 + 0];
							temp_bv[1] = sm_a[ll][idx_h1 + (idx_h3) * FUSION_SIZE_SLICE_2_H1 + 16];
							temp_bv[2] = sm_a[ll][idx_h1 + (idx_h3) * FUSION_SIZE_SLICE_2_H1 + 32];
							temp_bv[3] = sm_a[ll][idx_h1 + (idx_h3) * FUSION_SIZE_SLICE_2_H1 + 48];
				
							for (int xx = 0 ; xx < 4; xx++)
							{
								temp_av = sm_b[ll][idx_h2 + (idx_p6) * FUSION_SIZE_SLICE_2_H2 + (xx * 16)];
				
								reg_tile[0][xx] += temp_av * temp_bv[0];
								reg_tile[1][xx] += temp_av * temp_bv[1];
								reg_tile[2][xx] += temp_av * temp_bv[2];
								reg_tile[3][xx] += temp_av * temp_bv[3];
							}
						}
						__syncthreads();
					}
				}
			}
		}
	}
	

	// 
	//  register_transpose (top - bottom)
	//
	{
		if (threadIdx.y < 4)						// 0, 1, 2, 3
		{
			// sm_a[16][64] <-- (4 x 16) x (4 x 4) = (16 x 64)		  'y''x'
			sm_a[0 + threadIdx.y * 4][threadIdx.x] 			= reg_tile[0][0];
			sm_a[1 + threadIdx.y * 4][threadIdx.x] 			= reg_tile[1][0];
			sm_a[2 + threadIdx.y * 4][threadIdx.x] 			= reg_tile[2][0];
			sm_a[3 + threadIdx.y * 4][threadIdx.x] 			= reg_tile[3][0];
			
			sm_a[0 + threadIdx.y * 4][threadIdx.x + 16] 	= reg_tile[0][1];
			sm_a[1 + threadIdx.y * 4][threadIdx.x + 16] 	= reg_tile[1][1];
			sm_a[2 + threadIdx.y * 4][threadIdx.x + 16] 	= reg_tile[2][1];
			sm_a[3 + threadIdx.y * 4][threadIdx.x + 16] 	= reg_tile[3][1];
					
			sm_a[0 + threadIdx.y * 4][threadIdx.x + 32] 	= reg_tile[0][2];
			sm_a[1 + threadIdx.y * 4][threadIdx.x + 32] 	= reg_tile[1][2];
			sm_a[2 + threadIdx.y * 4][threadIdx.x + 32] 	= reg_tile[2][2];
			sm_a[3 + threadIdx.y * 4][threadIdx.x + 32] 	= reg_tile[3][2];
			
			sm_a[0 + threadIdx.y * 4][threadIdx.x + 48] 	= reg_tile[0][3];
			sm_a[1 + threadIdx.y * 4][threadIdx.x + 48] 	= reg_tile[1][3];
			sm_a[2 + threadIdx.y * 4][threadIdx.x + 48] 	= reg_tile[2][3];
			sm_a[3 + threadIdx.y * 4][threadIdx.x + 48] 	= reg_tile[3][3];
		}

		if (threadIdx.y >= 4 && threadIdx.y < 8)	// 4, 5, 6, 7
		{
			sm_b[0 + (threadIdx.y - 4) * 4][threadIdx.x] 		= reg_tile[0][0];
			sm_b[1 + (threadIdx.y - 4) * 4][threadIdx.x] 		= reg_tile[1][0];
			sm_b[2 + (threadIdx.y - 4) * 4][threadIdx.x] 		= reg_tile[2][0];
			sm_b[3 + (threadIdx.y - 4) * 4][threadIdx.x] 		= reg_tile[3][0];

			sm_b[0 + (threadIdx.y - 4) * 4][threadIdx.x + 16] 	= reg_tile[0][1];
			sm_b[1 + (threadIdx.y - 4) * 4][threadIdx.x + 16] 	= reg_tile[1][1];
			sm_b[2 + (threadIdx.y - 4) * 4][threadIdx.x + 16] 	= reg_tile[2][1];
			sm_b[3 + (threadIdx.y - 4) * 4][threadIdx.x + 16] 	= reg_tile[3][1];

			sm_b[0 + (threadIdx.y - 4) * 4][threadIdx.x + 32] 	= reg_tile[0][2];
			sm_b[1 + (threadIdx.y - 4) * 4][threadIdx.x + 32] 	= reg_tile[1][2];
			sm_b[2 + (threadIdx.y - 4) * 4][threadIdx.x + 32] 	= reg_tile[2][2];
			sm_b[3 + (threadIdx.y - 4) * 4][threadIdx.x + 32] 	= reg_tile[3][2];

			sm_b[0 + (threadIdx.y - 4) * 4][threadIdx.x + 48] 	= reg_tile[0][3];
			sm_b[1 + (threadIdx.y - 4) * 4][threadIdx.x + 48] 	= reg_tile[1][3];
			sm_b[2 + (threadIdx.y - 4) * 4][threadIdx.x + 48] 	= reg_tile[2][3];
			sm_b[3 + (threadIdx.y - 4) * 4][threadIdx.x + 48] 	= reg_tile[3][3];
		}
		__syncthreads();

		if (threadIdx.y < 4)						// 0, 1, 2, 3
		{
			reg_tile[0][0] = sm_a[threadIdx.y + 0][(threadIdx.x)];
			reg_tile[1][0] = sm_a[threadIdx.y + 4][(threadIdx.x)];
			reg_tile[2][0] = sm_a[threadIdx.y + 8][(threadIdx.x)];
			reg_tile[3][0] = sm_a[threadIdx.y + 12][(threadIdx.x)];

			reg_tile[0][1] = sm_a[threadIdx.y + 0][(threadIdx.x) + 16];
			reg_tile[1][1] = sm_a[threadIdx.y + 4][(threadIdx.x) + 16];
			reg_tile[2][1] = sm_a[threadIdx.y + 8][(threadIdx.x) + 16];
			reg_tile[3][1] = sm_a[threadIdx.y + 12][(threadIdx.x) + 16];
			
			reg_tile[0][2] = sm_a[threadIdx.y + 0][(threadIdx.x) + 32];
			reg_tile[1][2] = sm_a[threadIdx.y + 4][(threadIdx.x) + 32];
			reg_tile[2][2] = sm_a[threadIdx.y + 8][(threadIdx.x) + 32];
			reg_tile[3][2] = sm_a[threadIdx.y + 12][(threadIdx.x) + 32];
			
			reg_tile[0][3] = sm_a[threadIdx.y + 0][(threadIdx.x) + 48];
			reg_tile[1][3] = sm_a[threadIdx.y + 4][(threadIdx.x) + 48];
			reg_tile[2][3] = sm_a[threadIdx.y + 8][(threadIdx.x) + 48];
			reg_tile[3][3] = sm_a[threadIdx.y + 12][(threadIdx.x) + 48];
		}

		if (threadIdx.y >= 4 && threadIdx.y < 8)	// 4, 5, 6, 7
		{
			reg_tile[0][0] = sm_b[(threadIdx.y - 4) + 0][(threadIdx.x)];
			reg_tile[1][0] = sm_b[(threadIdx.y - 4) + 4][(threadIdx.x)];
			reg_tile[2][0] = sm_b[(threadIdx.y - 4) + 8][(threadIdx.x)];
			reg_tile[3][0] = sm_b[(threadIdx.y - 4) + 12][(threadIdx.x)];

			reg_tile[0][1] = sm_b[(threadIdx.y - 4) + 0][(threadIdx.x) + 16];
			reg_tile[1][1] = sm_b[(threadIdx.y - 4) + 4][(threadIdx.x) + 16];
			reg_tile[2][1] = sm_b[(threadIdx.y - 4) + 8][(threadIdx.x) + 16];
			reg_tile[3][1] = sm_b[(threadIdx.y - 4) + 12][(threadIdx.x) + 16];
			
			reg_tile[0][2] = sm_b[(threadIdx.y - 4) + 0][(threadIdx.x) + 32];
			reg_tile[1][2] = sm_b[(threadIdx.y - 4) + 4][(threadIdx.x) + 32];
			reg_tile[2][2] = sm_b[(threadIdx.y - 4) + 8][(threadIdx.x) + 32];
			reg_tile[3][2] = sm_b[(threadIdx.y - 4) + 12][(threadIdx.x) + 32];
			
			reg_tile[0][3] = sm_b[(threadIdx.y - 4) + 0][(threadIdx.x) + 48];	
			reg_tile[1][3] = sm_b[(threadIdx.y - 4) + 4][(threadIdx.x) + 48];
			reg_tile[2][3] = sm_b[(threadIdx.y - 4) + 8][(threadIdx.x) + 48];
			reg_tile[3][3] = sm_b[(threadIdx.y - 4) + 12][(threadIdx.x) + 48];
		}
		__syncthreads();

		if (threadIdx.y >= 8 && threadIdx.y < 12)	// 8, 9, 10, 11
		{
			sm_a[0 + (threadIdx.y - 8) * 4][threadIdx.x] = reg_tile[0][0];
			sm_a[1 + (threadIdx.y - 8) * 4][threadIdx.x] = reg_tile[1][0];
			sm_a[2 + (threadIdx.y - 8) * 4][threadIdx.x] = reg_tile[2][0];
			sm_a[3 + (threadIdx.y - 8) * 4][threadIdx.x] = reg_tile[3][0];

			sm_a[0 + (threadIdx.y - 8) * 4][threadIdx.x + 16] = reg_tile[0][1];
			sm_a[1 + (threadIdx.y - 8) * 4][threadIdx.x + 16] = reg_tile[1][1];
			sm_a[2 + (threadIdx.y - 8) * 4][threadIdx.x + 16] = reg_tile[2][1];
			sm_a[3 + (threadIdx.y - 8) * 4][threadIdx.x + 16] = reg_tile[3][1];

			sm_a[0 + (threadIdx.y - 8) * 4][threadIdx.x + 32] = reg_tile[0][2];
			sm_a[1 + (threadIdx.y - 8) * 4][threadIdx.x + 32] = reg_tile[1][2];
			sm_a[2 + (threadIdx.y - 8) * 4][threadIdx.x + 32] = reg_tile[2][2];
			sm_a[3 + (threadIdx.y - 8) * 4][threadIdx.x + 32] = reg_tile[3][2];

			sm_a[0 + (threadIdx.y - 8) * 4][threadIdx.x + 48] = reg_tile[0][3];
			sm_a[1 + (threadIdx.y - 8) * 4][threadIdx.x + 48] = reg_tile[1][3];
			sm_a[2 + (threadIdx.y - 8) * 4][threadIdx.x + 48] = reg_tile[2][3];
			sm_a[3 + (threadIdx.y - 8) * 4][threadIdx.x + 48] = reg_tile[3][3];
		}

		if (threadIdx.y >= 12)	// 12, 13, 14, 15
		{
			sm_b[0 + (threadIdx.y - 12) * 4][threadIdx.x] = reg_tile[0][0];
			sm_b[1 + (threadIdx.y - 12) * 4][threadIdx.x] = reg_tile[1][0];
			sm_b[2 + (threadIdx.y - 12) * 4][threadIdx.x] = reg_tile[2][0];
			sm_b[3 + (threadIdx.y - 12) * 4][threadIdx.x] = reg_tile[3][0];
			
			sm_b[0 + (threadIdx.y - 12) * 4][threadIdx.x + 16] = reg_tile[0][1];
			sm_b[1 + (threadIdx.y - 12) * 4][threadIdx.x + 16] = reg_tile[1][1];
			sm_b[2 + (threadIdx.y - 12) * 4][threadIdx.x + 16] = reg_tile[2][1];
			sm_b[3 + (threadIdx.y - 12) * 4][threadIdx.x + 16] = reg_tile[3][1];
			
			sm_b[0 + (threadIdx.y - 12) * 4][threadIdx.x + 32] = reg_tile[0][2];
			sm_b[1 + (threadIdx.y - 12) * 4][threadIdx.x + 32] = reg_tile[1][2];
			sm_b[2 + (threadIdx.y - 12) * 4][threadIdx.x + 32] = reg_tile[2][2];
			sm_b[3 + (threadIdx.y - 12) * 4][threadIdx.x + 32] = reg_tile[3][2];
			
			sm_b[0 + (threadIdx.y - 12) * 4][threadIdx.x + 48] = reg_tile[0][3];
			sm_b[1 + (threadIdx.y - 12) * 4][threadIdx.x + 48] = reg_tile[1][3];
			sm_b[2 + (threadIdx.y - 12) * 4][threadIdx.x + 48] = reg_tile[2][3];
			sm_b[3 + (threadIdx.y - 12) * 4][threadIdx.x + 48] = reg_tile[3][3];
		}
		__syncthreads();

		if (threadIdx.y >= 8 && threadIdx.y < 12)	// 8, 9, 10, 11
		{
			reg_tile[0][0] = sm_a[(threadIdx.y - 8) + 0][(threadIdx.x)];
			reg_tile[1][0] = sm_a[(threadIdx.y - 8) + 4][(threadIdx.x)];
			reg_tile[2][0] = sm_a[(threadIdx.y - 8) + 8][(threadIdx.x)];
			reg_tile[3][0] = sm_a[(threadIdx.y - 8) + 12][(threadIdx.x)];

			reg_tile[0][1] = sm_a[(threadIdx.y - 8) + 0][(threadIdx.x) + 16];
			reg_tile[1][1] = sm_a[(threadIdx.y - 8) + 4][(threadIdx.x) + 16];
			reg_tile[2][1] = sm_a[(threadIdx.y - 8) + 8][(threadIdx.x) + 16];
			reg_tile[3][1] = sm_a[(threadIdx.y - 8) + 12][(threadIdx.x) + 16];

			reg_tile[0][2] = sm_a[(threadIdx.y - 8) + 0][(threadIdx.x) + 32];
			reg_tile[1][2] = sm_a[(threadIdx.y - 8) + 4][(threadIdx.x) + 32];		
			reg_tile[2][2] = sm_a[(threadIdx.y - 8) + 8][(threadIdx.x) + 32];
			reg_tile[3][2] = sm_a[(threadIdx.y - 8) + 12][(threadIdx.x) + 32];

			reg_tile[0][3] = sm_a[(threadIdx.y - 8) + 0][(threadIdx.x) + 48];
			reg_tile[1][3] = sm_a[(threadIdx.y - 8) + 4][(threadIdx.x) + 48];
			reg_tile[2][3] = sm_a[(threadIdx.y - 8) + 8][(threadIdx.x) + 48];
			reg_tile[3][3] = sm_a[(threadIdx.y - 8) + 12][(threadIdx.x) + 48];
		}

		if (threadIdx.y >= 12)	// 12, 13, 14, 15
		{
			reg_tile[0][0] = sm_b[(threadIdx.y - 12) + 0][(threadIdx.x)];
			reg_tile[1][0] = sm_b[(threadIdx.y - 12) + 4][(threadIdx.x)];
			reg_tile[2][0] = sm_b[(threadIdx.y - 12) + 8][(threadIdx.x)];
			reg_tile[3][0] = sm_b[(threadIdx.y - 12) + 12][(threadIdx.x)];

			reg_tile[0][1] = sm_b[(threadIdx.y - 12) + 0][(threadIdx.x) + 16];
			reg_tile[1][1] = sm_b[(threadIdx.y - 12) + 4][(threadIdx.x) + 16];
			reg_tile[2][1] = sm_b[(threadIdx.y - 12) + 8][(threadIdx.x) + 16];
			reg_tile[3][1] = sm_b[(threadIdx.y - 12) + 12][(threadIdx.x) + 16];

			reg_tile[0][2] = sm_b[(threadIdx.y - 12) + 0][(threadIdx.x) + 32];
			reg_tile[1][2] = sm_b[(threadIdx.y - 12) + 4][(threadIdx.x) + 32];
			reg_tile[2][2] = sm_b[(threadIdx.y - 12) + 8][(threadIdx.x) + 32];
			reg_tile[3][2] = sm_b[(threadIdx.y - 12) + 12][(threadIdx.x) + 32];

			reg_tile[0][3] = sm_b[(threadIdx.y - 12) + 0][(threadIdx.x) + 48];
			reg_tile[1][3] = sm_b[(threadIdx.y - 12) + 4][(threadIdx.x) + 48];
			reg_tile[2][3] = sm_b[(threadIdx.y - 12) + 8][(threadIdx.x) + 48];
			reg_tile[3][3] = sm_b[(threadIdx.y - 12) + 12][(threadIdx.x) + 48];
		}
		__syncthreads();
	}


	// 
	#pragma unroll 1
	for (int iter_ia6 = 0; iter_ia6 < NUM_IA6_LOOPS; iter_ia6++)
	{
		
		//  doubles (d1 and d2) 
		{
		// #if 0
			//  d1-bottom: sd1_4, 5 , 6 , 7 , 8 and 9.
			#pragma unroll 1
			for (int iter_noab = 0; iter_noab < size_noab; iter_noab++)
			{
				// 	flags
				int flag_d1_4 = const_list_d1_flags_offset[3 + (iter_noab + (iter_ia6) * size_noab) * NUM_D1_EQUATIONS];
				int flag_d1_5 = const_list_d1_flags_offset[4 + (iter_noab + (iter_ia6) * size_noab) * NUM_D1_EQUATIONS];
				int flag_d1_6 = const_list_d1_flags_offset[5 + (iter_noab + (iter_ia6) * size_noab) * NUM_D1_EQUATIONS];
				int flag_d1_7 = const_list_d1_flags_offset[6 + (iter_noab + (iter_ia6) * size_noab) * NUM_D1_EQUATIONS];
				int flag_d1_8 = const_list_d1_flags_offset[7 + (iter_noab + (iter_ia6) * size_noab) * NUM_D1_EQUATIONS];
				int flag_d1_9 = const_list_d1_flags_offset[8 + (iter_noab + (iter_ia6) * size_noab) * NUM_D1_EQUATIONS];

				// 
				// int local_d1_size_idx_h1b = const_list_d1_problem_size[0 + (iter_noab + (iter_ia6) * size_noab) * NUM_D1_INDEX];
				// int local_d1_size_idx_h2b = const_list_d1_problem_size[1 + (iter_noab + (iter_ia6) * size_noab) * NUM_D1_INDEX];
				// int local_d1_size_idx_h3b = const_list_d1_problem_size[2 + (iter_noab + (iter_ia6) * size_noab) * NUM_D1_INDEX];
				// // int local_d1_size_idx_h7b = const_list_d1_problem_size[3 + (iter_noab + (iter_ia6) * size_noab) * NUM_D1_INDEX];
				// int local_d1_size_idx_p4b = const_list_d1_problem_size[4 + (iter_noab + (iter_ia6) * size_noab) * NUM_D1_INDEX];
				// int local_d1_size_idx_p5b = const_list_d1_problem_size[5 + (iter_noab + (iter_ia6) * size_noab) * NUM_D1_INDEX];
				// int local_d1_size_idx_p6b = const_list_d1_problem_size[6 + (iter_noab + (iter_ia6) * size_noab) * NUM_D1_INDEX];
				base_size_h1b = const_list_d1_problem_size[0 + (iter_noab + (iter_ia6) * size_noab) * NUM_D1_INDEX];
				base_size_h2b = const_list_d1_problem_size[1 + (iter_noab + (iter_ia6) * size_noab) * NUM_D1_INDEX];
				base_size_h3b = const_list_d1_problem_size[2 + (iter_noab + (iter_ia6) * size_noab) * NUM_D1_INDEX];
				base_size_h7b = const_list_d1_problem_size[3 + (iter_noab + (iter_ia6) * size_noab) * NUM_D1_INDEX];
				base_size_p4b = const_list_d1_problem_size[4 + (iter_noab + (iter_ia6) * size_noab) * NUM_D1_INDEX];
				base_size_p5b = const_list_d1_problem_size[5 + (iter_noab + (iter_ia6) * size_noab) * NUM_D1_INDEX];
				base_size_p6b = const_list_d1_problem_size[6 + (iter_noab + (iter_ia6) * size_noab) * NUM_D1_INDEX];

				// 
				// if (blockIdx.x == 0 && threadIdx.x == 0 && threadIdx.y == 0)
				// {
				// 	printf ("[sd1][part2][ia6 = %2d] h1,h2,h3,p4,p5,p6 = %2d,%2d,%2d,%2d,%2d,%2d\n", iter_ia6, base_size_h1b, base_size_h2b, base_size_h3b, base_size_p4b, base_size_p5b, base_size_p6b);
				// }

				//	otheres according to the above problem-sizes
				//	(1) num_blks_h/p*b
				// num_blks_h1b = CEIL(local_d1_size_idx_h1b, FUSION_SIZE_SLICE_1_H1);
				// num_blks_h2b = CEIL(local_d1_size_idx_h2b, FUSION_SIZE_SLICE_1_H2);
				// num_blks_h3b = CEIL(local_d1_size_idx_h3b, FUSION_SIZE_SLICE_1_H3);
				// num_blks_p4b = CEIL(local_d1_size_idx_p4b, FUSION_SIZE_SLICE_1_P4);
				// num_blks_p5b = CEIL(local_d1_size_idx_p5b, FUSION_SIZE_SLICE_1_P5);
				// num_blks_p6b = CEIL(local_d1_size_idx_p6b, FUSION_SIZE_SLICE_1_P6);
				num_blks_h1b = CEIL(base_size_h1b, FUSION_SIZE_SLICE_1_H1);
				num_blks_h2b = CEIL(base_size_h2b, FUSION_SIZE_SLICE_1_H2);
				num_blks_h3b = CEIL(base_size_h3b, FUSION_SIZE_SLICE_1_H3);
				num_blks_p4b = CEIL(base_size_p4b, FUSION_SIZE_SLICE_1_P4);
				num_blks_p5b = CEIL(base_size_p5b, FUSION_SIZE_SLICE_1_P5);
				num_blks_p6b = CEIL(base_size_p6b, FUSION_SIZE_SLICE_1_P6);

				// 	(2) blk_idx_h/p*b
				blk_idx_p4b     = blockIdx.x / (num_blks_h3b * num_blks_h2b * num_blks_h1b * num_blks_p6b * num_blks_p5b);
				tmp_blkIdx      = blockIdx.x % (num_blks_h3b * num_blks_h2b * num_blks_h1b * num_blks_p6b * num_blks_p5b);
				blk_idx_p5b     = (tmp_blkIdx) / (num_blks_h3b * num_blks_h2b * num_blks_h1b * num_blks_p6b);
				tmp_blkIdx  	= (tmp_blkIdx) % (num_blks_h3b * num_blks_h2b * num_blks_h1b * num_blks_p6b);
				blk_idx_p6b     = (tmp_blkIdx) / (num_blks_h3b * num_blks_h2b * num_blks_h1b);
				tmp_blkIdx 		= (tmp_blkIdx) % (num_blks_h3b * num_blks_h2b * num_blks_h1b);
				blk_idx_h1b     = (tmp_blkIdx) / (num_blks_h3b * num_blks_h2b);
				tmp_blkIdx 		= (tmp_blkIdx) % (num_blks_h3b * num_blks_h2b);
				blk_idx_h2b 	= (tmp_blkIdx) / (num_blks_h3b);
				blk_idx_h3b		= blockIdx.x % (num_blks_h3b);

				// 	(3) str_blk_idx_h/p*
				str_blk_idx_h3 	= blk_idx_h3b * FUSION_SIZE_SLICE_1_H3;
				str_blk_idx_h2 	= blk_idx_h2b * FUSION_SIZE_SLICE_1_H2;
				str_blk_idx_h1 	= blk_idx_h1b * FUSION_SIZE_SLICE_1_H1;
				str_blk_idx_p6 	= blk_idx_p6b * FUSION_SIZE_SLICE_1_P6;
				str_blk_idx_p5 	= blk_idx_p5b * FUSION_SIZE_SLICE_1_P5;
				str_blk_idx_p4 	= blk_idx_p4b * FUSION_SIZE_SLICE_1_P4;

				// 	(4) rng_h/p*
				if ((base_size_h3b - (str_blk_idx_h3)) >= FUSION_SIZE_SLICE_1_H3)
					rng_h3 = FUSION_SIZE_SLICE_1_H3;
				else
					rng_h3 = base_size_h3b % FUSION_SIZE_SLICE_1_H3;
				
				if ((base_size_h2b - (str_blk_idx_h2)) >= FUSION_SIZE_SLICE_1_H2)
					rng_h2 = FUSION_SIZE_SLICE_1_H2;
				else
					rng_h2 = base_size_h2b % FUSION_SIZE_SLICE_1_H2;

				if ((base_size_h1b - (str_blk_idx_h1)) >= FUSION_SIZE_SLICE_1_H1)
					rng_h1 = FUSION_SIZE_SLICE_1_H1;
				else
					rng_h1 = base_size_h1b % FUSION_SIZE_SLICE_1_H1;
				
				if ((base_size_p6b - (str_blk_idx_p6)) >= FUSION_SIZE_SLICE_1_P6)
					rng_p6 = FUSION_SIZE_SLICE_1_P6;
				else
					rng_p6 = base_size_p6b % FUSION_SIZE_SLICE_1_P6;

				if ((base_size_p5b - (str_blk_idx_p5)) >= FUSION_SIZE_SLICE_1_P5)
					rng_p5 = FUSION_SIZE_SLICE_1_P5;
				else
					rng_p5 = base_size_p5b % FUSION_SIZE_SLICE_1_P5;

				if ((base_size_p4b - (str_blk_idx_p4)) >= FUSION_SIZE_SLICE_1_P4)
					rng_p4 = FUSION_SIZE_SLICE_1_P4;
				else
					rng_p4 = base_size_p4b % FUSION_SIZE_SLICE_1_P4;

				// 	sd1_4
				if (flag_d1_4 >= 0)
				{
					// 
					double* tmp_dev_d1_t2_4 = dev_d1_t2_all + size_max_dim_d1_t2 * flag_d1_4;
					double* tmp_dev_d1_v2_4 = dev_d1_v2_all + size_max_dim_d1_v2 * flag_d1_4;

					internal_upperbound = 0;
					#pragma unroll 1
					for (int l = 0; l < base_size_h7b; l+= FUSION_SIZE_INT_UNIT)
					{
						// Part: Generalized Contraction Index (p7b)
						internal_offset = (l + FUSION_SIZE_INT_UNIT) - base_size_h7b;
						if (internal_offset > 0) internal_upperbound = internal_offset;

						// Load Input Tensor to Shared Memory: 16:16
						// # of size_internal Indices: 1
						if (idx_p6 < rng_p6 && idx_h1 < rng_h1 && threadIdx.x < FUSION_SIZE_INT_UNIT - internal_upperbound)
						for (int ll = 0; ll < rng_p5; ll++)
						{
							sm_a[threadIdx.x][threadIdx.y + ll * FUSION_SIZE_TB_1_Y] = tmp_dev_d1_t2_4[(str_blk_idx_p5 + ll + (str_blk_idx_p6 + idx_p6 + (str_blk_idx_h1 + idx_h1) * base_size_p6b) * base_size_p5b) * base_size_h7b + (threadIdx.x + l)];					
						}

						// Load Input Tensor to Shared Memory
						if (idx_h3 < rng_h3 && idx_h2 < rng_h2 && threadIdx.y < FUSION_SIZE_INT_UNIT - internal_upperbound)
						for (int ll = 0; ll < rng_p4; ll++)
						{
							sm_b[threadIdx.y][threadIdx.x + ll * FUSION_SIZE_TB_1_X] = tmp_dev_d1_v2_4[(str_blk_idx_h3 + idx_h3 + (str_blk_idx_h2 + idx_h2 + (str_blk_idx_p4 + ll + (threadIdx.y + l) * base_size_p4b) * base_size_h2b) * base_size_h3b)];
						}
						__syncthreads();

						// Cross-Product: -1
						// Part: Generalized Threads
						for (int ll = 0; ll < FUSION_SIZE_INT_UNIT - internal_upperbound; ll++)
						{
							temp_bv[0] = sm_b[ll][idx_h3 + (idx_h2) * FUSION_SIZE_SLICE_1_H3 + 0];
							temp_bv[1] = sm_b[ll][idx_h3 + (idx_h2) * FUSION_SIZE_SLICE_1_H3 + 16];
							temp_bv[2] = sm_b[ll][idx_h3 + (idx_h2) * FUSION_SIZE_SLICE_1_H3 + 32];
							temp_bv[3] = sm_b[ll][idx_h3 + (idx_h2) * FUSION_SIZE_SLICE_1_H3 + 48];

							for (int xx = 0 ; xx < 4; xx++)
							{
								temp_av = sm_a[ll][idx_p6 + (idx_h1) * FUSION_SIZE_SLICE_1_P6 + (xx * 16)];

								reg_tile[0][xx] -= temp_av * temp_bv[0];
								reg_tile[1][xx] -= temp_av * temp_bv[1];
								reg_tile[2][xx] -= temp_av * temp_bv[2];
								reg_tile[3][xx] -= temp_av * temp_bv[3];
							}
						}
						__syncthreads();
					}
				}

				// 	sd1_5
				if (flag_d1_5 >= 0)
				{
					// 
					double* tmp_dev_d1_t2_5 = dev_d1_t2_all + size_max_dim_d1_t2 * flag_d1_5;
					double* tmp_dev_d1_v2_5 = dev_d1_v2_all + size_max_dim_d1_v2 * flag_d1_5;

					internal_upperbound = 0;
					#pragma unroll 1
					for (int l = 0; l < base_size_h7b; l+= FUSION_SIZE_INT_UNIT)
					{
						// Part: Generalized Contraction Index (p7b)
						// internal_offset = (l + FUSION_SIZE_INT_UNIT) - size_internal;
						internal_offset = (l + FUSION_SIZE_INT_UNIT) - base_size_h7b;
						if (internal_offset > 0) internal_upperbound = internal_offset;

						// Load Input Tensor to Shared Memory: 16:16
						// # of Internal Indices: 1
						if (idx_p6 < rng_p6 && idx_h1 < rng_h2 && threadIdx.x < FUSION_SIZE_INT_UNIT - internal_upperbound)
						for (int ll = 0; ll < rng_p5; ll++)
						{
							sm_a[threadIdx.x][threadIdx.y + ll * FUSION_SIZE_TB_1_Y] = tmp_dev_d1_t2_5[(str_blk_idx_p5 + ll + (str_blk_idx_p6 + idx_p6 + (str_blk_idx_h2 + idx_h1) * base_size_p6b) * base_size_p5b) * base_size_h7b + (threadIdx.x + l)]; 
						}

						// Load Input Tensor to Shared Memory
						if (idx_h3 < rng_h3 && idx_h2 < rng_h1 && threadIdx.y < FUSION_SIZE_INT_UNIT - internal_upperbound)
						for (int ll = 0; ll < rng_p4; ll++)
						{
							sm_b[threadIdx.y][threadIdx.x + ll * FUSION_SIZE_TB_1_X] = tmp_dev_d1_v2_5[(str_blk_idx_h3 + idx_h3 + (str_blk_idx_h1 + idx_h2 + (str_blk_idx_p4 + ll + (threadIdx.y + l) * base_size_p4b) * base_size_h1b) * base_size_h3b)];
						}
						__syncthreads();

						// Cross-Product: -1
						// Part: Generalized Threads
						for (int ll = 0; ll < FUSION_SIZE_INT_UNIT - internal_upperbound; ll++)
						{
							temp_bv[0] = sm_b[ll][idx_h3 + (idx_h1) * FUSION_SIZE_SLICE_1_H3 + 0];
							temp_bv[1] = sm_b[ll][idx_h3 + (idx_h1) * FUSION_SIZE_SLICE_1_H3 + 16];
							temp_bv[2] = sm_b[ll][idx_h3 + (idx_h1) * FUSION_SIZE_SLICE_1_H3 + 32];
							temp_bv[3] = sm_b[ll][idx_h3 + (idx_h1) * FUSION_SIZE_SLICE_1_H3 + 48];

							for (int xx = 0 ; xx < 4; xx++)
							{
								temp_av = sm_a[ll][idx_p6 + (idx_h2) * FUSION_SIZE_SLICE_1_P6 + (xx * 16)];

								reg_tile[0][xx] += temp_av * temp_bv[0];
								reg_tile[1][xx] += temp_av * temp_bv[1];
								reg_tile[2][xx] += temp_av * temp_bv[2];
								reg_tile[3][xx] += temp_av * temp_bv[3];
							}
						}
						__syncthreads();
					}
				}

				// 	sd1_6
				if (flag_d1_6 >= 0)
				{
					// 
					double* tmp_dev_d1_t2_6 = dev_d1_t2_all + size_max_dim_d1_t2 * flag_d1_6;
					double* tmp_dev_d1_v2_6 = dev_d1_v2_all + size_max_dim_d1_v2 * flag_d1_6;

					internal_upperbound = 0;
					#pragma unroll 1
					for (int l = 0; l < base_size_h7b; l+= FUSION_SIZE_INT_UNIT)
					{
						// Part: Generalized Contraction Index (p7b)
						// internal_offset = (l + FUSION_SIZE_INT_UNIT) - size_internal;
						internal_offset = (l + FUSION_SIZE_INT_UNIT) - base_size_h7b;
						if (internal_offset > 0) internal_upperbound = internal_offset;

						// Load Input Tensor to Shared Memory: 16:16
						// # of Internal Indices: 1 //63, 21
						if (idx_p6 < rng_p6 && idx_h1 < rng_h3 && threadIdx.x < FUSION_SIZE_INT_UNIT - internal_upperbound)
						for (int ll = 0; ll < rng_p5; ll++)
						{
							sm_a[threadIdx.x][threadIdx.y + ll * FUSION_SIZE_TB_1_Y] = tmp_dev_d1_t2_6[(str_blk_idx_p5 + ll + (str_blk_idx_p6 + idx_p6 + (str_blk_idx_h3 + idx_h1) * base_size_p6b) * base_size_p5b) * base_size_h7b + (threadIdx.x + l)];
						}

						// Load Input Tensor to Shared Memory
						if (idx_h3 < rng_h2 && idx_h2 < rng_h1 && threadIdx.y < FUSION_SIZE_INT_UNIT - internal_upperbound)
						for (int ll = 0; ll < rng_p4; ll++)
						{
							sm_b[threadIdx.y][threadIdx.x + ll * FUSION_SIZE_TB_1_X] = tmp_dev_d1_v2_6[(str_blk_idx_h2 + idx_h3 + (str_blk_idx_h1 + idx_h2 + (str_blk_idx_p4 + ll + (threadIdx.y + l) * base_size_p4b) * base_size_h1b) * base_size_h2b)];
																										
						}
						__syncthreads();

						// Cross-Product: -1
						// Part: Generalized Threads
						for (int ll = 0; ll < FUSION_SIZE_INT_UNIT - internal_upperbound; ll++)
						{
							temp_bv[0] = sm_b[ll][idx_h2 + (idx_h1) * FUSION_SIZE_SLICE_1_H2 + 0];
							temp_bv[1] = sm_b[ll][idx_h2 + (idx_h1) * FUSION_SIZE_SLICE_1_H2 + 16];
							temp_bv[2] = sm_b[ll][idx_h2 + (idx_h1) * FUSION_SIZE_SLICE_1_H2 + 32];
							temp_bv[3] = sm_b[ll][idx_h2 + (idx_h1) * FUSION_SIZE_SLICE_1_H2 + 48];

							for (int xx = 0 ; xx < 4; xx++)
							{
								temp_av = sm_a[ll][idx_p6 + (idx_h3) * FUSION_SIZE_SLICE_1_P6 + (xx * 16)];

								reg_tile[0][xx] -= temp_av * temp_bv[0];
								reg_tile[1][xx] -= temp_av * temp_bv[1];
								reg_tile[2][xx] -= temp_av * temp_bv[2];
								reg_tile[3][xx] -= temp_av * temp_bv[3];
							}
						}
						__syncthreads();
					}
				}

				// 	sd1_7
				if (flag_d1_7 >= 0)
				{
					// 
					double* tmp_dev_d1_t2_7 = dev_d1_t2_all + size_max_dim_d1_t2 * flag_d1_7;
					double* tmp_dev_d1_v2_7 = dev_d1_v2_all + size_max_dim_d1_v2 * flag_d1_7;

					internal_upperbound = 0;
					#pragma unroll 1
					for (int l = 0; l < base_size_h7b; l+= FUSION_SIZE_INT_UNIT)
					{
						// Part: Generalized Contraction Index (p7b)
						internal_offset = (l + FUSION_SIZE_INT_UNIT) - base_size_h7b;
						if (internal_offset > 0) internal_upperbound = internal_offset;

						// Load Input Tensor to Shared Memory: 16:16
						// # of Internal Indices: 1
						if (idx_p6 < rng_p6 && idx_h1 < rng_h1 && threadIdx.x < FUSION_SIZE_INT_UNIT - internal_upperbound)
						for (int ll = 0; ll < rng_p4; ll++)
						{
							sm_a[threadIdx.x][threadIdx.y + ll * FUSION_SIZE_TB_1_Y] = tmp_dev_d1_t2_7[(str_blk_idx_p4 + ll + (str_blk_idx_p6 + idx_p6 + (str_blk_idx_h1 + idx_h1) * base_size_p6b) * base_size_p4b) * base_size_h7b + (threadIdx.x + l)];
						}

						// Load Input Tensor to Shared Memory
						if (idx_h3 < rng_h3 && idx_h2 < rng_h2 && threadIdx.y < FUSION_SIZE_INT_UNIT - internal_upperbound)
						for (int ll = 0; ll < rng_p5; ll++)
						{
							sm_b[threadIdx.y][threadIdx.x + ll * FUSION_SIZE_TB_1_X] = tmp_dev_d1_v2_7[(str_blk_idx_h3 + idx_h3 + (str_blk_idx_h2 + idx_h2 + (str_blk_idx_p5 + ll + (threadIdx.y + l) * base_size_p5b) * base_size_h2b) * base_size_h3b)];
						}
						__syncthreads();

						// Cross-Product: -1
						// Part: Generalized Threads
						for (int ll = 0; ll < FUSION_SIZE_INT_UNIT - internal_upperbound; ll++)
						{
							temp_bv[0] = sm_a[ll][idx_p6 + (idx_h1) * FUSION_SIZE_SLICE_1_P6 + 0];
							temp_bv[1] = sm_a[ll][idx_p6 + (idx_h1) * FUSION_SIZE_SLICE_1_P6 + 16];
							temp_bv[2] = sm_a[ll][idx_p6 + (idx_h1) * FUSION_SIZE_SLICE_1_P6 + 32];
							temp_bv[3] = sm_a[ll][idx_p6 + (idx_h1) * FUSION_SIZE_SLICE_1_P6 + 48];

							for (int xx = 0 ; xx < 4; xx++)
							{
								temp_av = sm_b[ll][idx_h3 + (idx_h2) * FUSION_SIZE_SLICE_1_H3 + (xx * 16)];

								reg_tile[0][xx] += temp_av * temp_bv[0];
								reg_tile[1][xx] += temp_av * temp_bv[1];
								reg_tile[2][xx] += temp_av * temp_bv[2];
								reg_tile[3][xx] += temp_av * temp_bv[3];
							}
						}
						__syncthreads();
					}
				}
				
				// 	sd1_8
				if (flag_d1_8 >= 0)
				{
					// 
					double* tmp_dev_d1_t2_8 = dev_d1_t2_all + size_max_dim_d1_t2 * flag_d1_8;
					double* tmp_dev_d1_v2_8 = dev_d1_v2_all + size_max_dim_d1_v2 * flag_d1_8;

					internal_upperbound = 0;
					#pragma unroll 1
					for (int l = 0; l < base_size_h7b; l+= FUSION_SIZE_INT_UNIT)
					{
						// Part: Generalized Contraction Index (p7b)
						internal_offset = (l + FUSION_SIZE_INT_UNIT) - base_size_h7b;
						if (internal_offset > 0) internal_upperbound = internal_offset;

						// Load Input Tensor to Shared Memory: 16:16
						// # of Internal Indices: 1
						if (idx_p6 < rng_p6 && idx_h1 < rng_h2 && threadIdx.x < FUSION_SIZE_INT_UNIT - internal_upperbound)
						for (int ll = 0; ll < rng_p4; ll++)
						{
							sm_a[threadIdx.x][threadIdx.y + ll * FUSION_SIZE_TB_1_Y] = tmp_dev_d1_t2_8[(str_blk_idx_p4 + ll + (str_blk_idx_p6 + idx_p6 + (str_blk_idx_h2 + idx_h1) * base_size_p6b) * base_size_p4b) * base_size_h7b + (threadIdx.x + l)];
						}

						// Load Input Tensor to Shared Memory
						if (idx_h3 < rng_h3 && idx_h2 < rng_h1 && threadIdx.y < FUSION_SIZE_INT_UNIT - internal_upperbound)
						for (int ll = 0; ll < rng_p5; ll++)
						{
							sm_b[threadIdx.y][threadIdx.x + ll * FUSION_SIZE_TB_1_X] = tmp_dev_d1_v2_8[(str_blk_idx_h3 + idx_h3 + (str_blk_idx_h1 + idx_h2 + (str_blk_idx_p5 + ll + (threadIdx.y + l) * base_size_p5b) * base_size_h1b) * base_size_h3b)];
						}
						__syncthreads();

						// Cross-Product: -1
						// Part: Generalized Threads
						for (int ll = 0; ll < FUSION_SIZE_INT_UNIT - internal_upperbound; ll++)
						{
							temp_bv[0] = sm_a[ll][idx_p6 + (idx_h2) * FUSION_SIZE_SLICE_1_P6 + 0];
							temp_bv[1] = sm_a[ll][idx_p6 + (idx_h2) * FUSION_SIZE_SLICE_1_P6 + 16];
							temp_bv[2] = sm_a[ll][idx_p6 + (idx_h2) * FUSION_SIZE_SLICE_1_P6 + 32];
							temp_bv[3] = sm_a[ll][idx_p6 + (idx_h2) * FUSION_SIZE_SLICE_1_P6 + 48];

							for (int xx = 0 ; xx < 4; xx++)
							{
								temp_av = sm_b[ll][idx_h3 + (idx_h1) * FUSION_SIZE_SLICE_1_H3 + (xx * 16)];

								reg_tile[0][xx] -= temp_av * temp_bv[0];
								reg_tile[1][xx] -= temp_av * temp_bv[1];
								reg_tile[2][xx] -= temp_av * temp_bv[2];
								reg_tile[3][xx] -= temp_av * temp_bv[3];
							}
						}
						__syncthreads();
					}
				}

				// 	sd1_9
				if (flag_d1_9 >= 0)
				{
					// 
					double* tmp_dev_d1_t2_9 = dev_d1_t2_all + size_max_dim_d1_t2 * flag_d1_9;
					double* tmp_dev_d1_v2_9 = dev_d1_v2_all + size_max_dim_d1_v2 * flag_d1_9;

					internal_upperbound = 0;
					#pragma unroll 1
					for (int l = 0; l < base_size_h7b; l+= FUSION_SIZE_INT_UNIT)
					{
						// Part: Generalized Contraction Index (p7b)
						internal_offset = (l + FUSION_SIZE_INT_UNIT) - base_size_h7b;
						if (internal_offset > 0) internal_upperbound = internal_offset;

						// Load Input Tensor to Shared Memory: 16:16
						// # of Internal Indices: 1
						if (idx_p6 < rng_p6 && idx_h1 < rng_h3 && threadIdx.x < FUSION_SIZE_INT_UNIT - internal_upperbound)
						for (int ll = 0; ll < rng_p4; ll++)
						{
							sm_a[threadIdx.x][threadIdx.y + ll * FUSION_SIZE_TB_1_Y] = tmp_dev_d1_t2_9[(str_blk_idx_p4 + ll + (str_blk_idx_p6 + idx_p6 + (str_blk_idx_h3 + idx_h1) * base_size_p6b) * base_size_p4b) * base_size_h7b + (threadIdx.x + l)];
						}

						// Load Input Tensor to Shared Memory
						if (idx_h3 < rng_h2 && idx_h2 < rng_h1 && threadIdx.y < FUSION_SIZE_INT_UNIT - internal_upperbound)
						for (int ll = 0; ll < rng_p5; ll++)
						{
							sm_b[threadIdx.y][threadIdx.x + ll * FUSION_SIZE_TB_1_X] = tmp_dev_d1_v2_9[(str_blk_idx_h2 + idx_h3 + (str_blk_idx_h1 + idx_h2 + (str_blk_idx_p5 + ll + (threadIdx.y + l) * base_size_p5b) * base_size_h1b) * base_size_h2b)];
						}
						__syncthreads();

						// Cross-Product: -1
						// Part: Generalized Threads
						for (int ll = 0; ll < FUSION_SIZE_INT_UNIT - internal_upperbound; ll++)
						{
							temp_bv[0] = sm_a[ll][idx_p6 + (idx_h3) * FUSION_SIZE_SLICE_1_P6 + 0];
							temp_bv[1] = sm_a[ll][idx_p6 + (idx_h3) * FUSION_SIZE_SLICE_1_P6 + 16];
							temp_bv[2] = sm_a[ll][idx_p6 + (idx_h3) * FUSION_SIZE_SLICE_1_P6 + 32];
							temp_bv[3] = sm_a[ll][idx_p6 + (idx_h3) * FUSION_SIZE_SLICE_1_P6 + 48];

							for (int xx = 0 ; xx < 4; xx++)
							{
								temp_av = sm_b[ll][idx_h2 + (idx_h1) * FUSION_SIZE_SLICE_1_H2 + (xx * 16)];

								reg_tile[0][xx] += temp_av * temp_bv[0];
								reg_tile[1][xx] += temp_av * temp_bv[1];
								reg_tile[2][xx] += temp_av * temp_bv[2];
								reg_tile[3][xx] += temp_av * temp_bv[3];
							}
						}
						__syncthreads();
					}
				}
			}
		
		
			//  d2-bottom: sd2_1, 2, 3, 4, 5 and 6.
			#pragma unroll 1
			for (int iter_nvab = 0; iter_nvab < size_nvab; iter_nvab++)
			{
				//
				int flag_d2_1 = const_list_d2_flags_offset[0 + (iter_nvab + (iter_ia6) * size_nvab) * NUM_D2_EQUATIONS];
				int flag_d2_2 = const_list_d2_flags_offset[1 + (iter_nvab + (iter_ia6) * size_nvab) * NUM_D2_EQUATIONS];
				int flag_d2_3 = const_list_d2_flags_offset[2 + (iter_nvab + (iter_ia6) * size_nvab) * NUM_D2_EQUATIONS];
				int flag_d2_4 = const_list_d2_flags_offset[3 + (iter_nvab + (iter_ia6) * size_nvab) * NUM_D2_EQUATIONS];
				int flag_d2_5 = const_list_d2_flags_offset[4 + (iter_nvab + (iter_ia6) * size_nvab) * NUM_D2_EQUATIONS];
				int flag_d2_6 = const_list_d2_flags_offset[5 + (iter_nvab + (iter_ia6) * size_nvab) * NUM_D2_EQUATIONS];

				// 
				// int local_d2_size_idx_h1b = const_list_d2_problem_size[0 + (iter_nvab + (iter_ia6) * size_nvab) * NUM_D2_INDEX];
				// int local_d2_size_idx_h2b = const_list_d2_problem_size[1 + (iter_nvab + (iter_ia6) * size_nvab) * NUM_D2_INDEX];
				// int local_d2_size_idx_h3b = const_list_d2_problem_size[2 + (iter_nvab + (iter_ia6) * size_nvab) * NUM_D2_INDEX];
				// int local_d2_size_idx_p4b = const_list_d2_problem_size[3 + (iter_nvab + (iter_ia6) * size_nvab) * NUM_D2_INDEX];
				// int local_d2_size_idx_p5b = const_list_d2_problem_size[4 + (iter_nvab + (iter_ia6) * size_nvab) * NUM_D2_INDEX];
				// int local_d2_size_idx_p6b = const_list_d2_problem_size[5 + (iter_nvab + (iter_ia6) * size_nvab) * NUM_D2_INDEX];
				// int local_d2_size_idx_p7b = const_list_d2_problem_size[6 + (iter_nvab + (iter_ia6) * size_nvab) * NUM_D2_INDEX];
				base_size_h1b = const_list_d2_problem_size[0 + (iter_nvab + (iter_ia6) * size_nvab) * NUM_D2_INDEX];
				base_size_h2b = const_list_d2_problem_size[1 + (iter_nvab + (iter_ia6) * size_nvab) * NUM_D2_INDEX];
				base_size_h3b = const_list_d2_problem_size[2 + (iter_nvab + (iter_ia6) * size_nvab) * NUM_D2_INDEX];
				base_size_p4b = const_list_d2_problem_size[3 + (iter_nvab + (iter_ia6) * size_nvab) * NUM_D2_INDEX];
				base_size_p5b = const_list_d2_problem_size[4 + (iter_nvab + (iter_ia6) * size_nvab) * NUM_D2_INDEX];
				base_size_p6b = const_list_d2_problem_size[5 + (iter_nvab + (iter_ia6) * size_nvab) * NUM_D2_INDEX];
				base_size_p7b = const_list_d2_problem_size[6 + (iter_nvab + (iter_ia6) * size_nvab) * NUM_D2_INDEX];

				// // 
				// if (blockIdx.x == 0 && threadIdx.x == 0 && threadIdx.y == 0)
				// {
				// 	printf ("[sd2][part2][ia6 = %2d] h1,h2,h3,p4,p5,p6 = %2d,%2d,%2d,%2d,%2d,%2d\n", iter_ia6, base_size_h1b, base_size_h2b, base_size_h3b, base_size_p4b, base_size_p5b, base_size_p6b);
				// }

				//	otheres according to the above problem-sizes
				//	(1) num_blks_h/p*b
				// num_blks_h1b = CEIL(local_d2_size_idx_h1b, FUSION_SIZE_SLICE_1_H1);
				// num_blks_h2b = CEIL(local_d2_size_idx_h2b, FUSION_SIZE_SLICE_1_H2);
				// num_blks_h3b = CEIL(local_d2_size_idx_h3b, FUSION_SIZE_SLICE_1_H3);
				// num_blks_p4b = CEIL(local_d2_size_idx_p4b, FUSION_SIZE_SLICE_1_P4);
				// num_blks_p5b = CEIL(local_d2_size_idx_p5b, FUSION_SIZE_SLICE_1_P5);
				// num_blks_p6b = CEIL(local_d2_size_idx_p6b, FUSION_SIZE_SLICE_1_P6);
				num_blks_h1b = CEIL(base_size_h1b, FUSION_SIZE_SLICE_1_H1);
				num_blks_h2b = CEIL(base_size_h2b, FUSION_SIZE_SLICE_1_H2);
				num_blks_h3b = CEIL(base_size_h3b, FUSION_SIZE_SLICE_1_H3);
				num_blks_p4b = CEIL(base_size_p4b, FUSION_SIZE_SLICE_1_P4);
				num_blks_p5b = CEIL(base_size_p5b, FUSION_SIZE_SLICE_1_P5);
				num_blks_p6b = CEIL(base_size_p6b, FUSION_SIZE_SLICE_1_P6);

				// 	(2) blk_idx_h/p*b
				blk_idx_p4b     = blockIdx.x / (num_blks_h3b * num_blks_h2b * num_blks_h1b * num_blks_p6b * num_blks_p5b);
				tmp_blkIdx      = blockIdx.x % (num_blks_h3b * num_blks_h2b * num_blks_h1b * num_blks_p6b * num_blks_p5b);
				blk_idx_p5b     = (tmp_blkIdx) / (num_blks_h3b * num_blks_h2b * num_blks_h1b * num_blks_p6b);
				tmp_blkIdx  	= (tmp_blkIdx) % (num_blks_h3b * num_blks_h2b * num_blks_h1b * num_blks_p6b);
				blk_idx_p6b     = (tmp_blkIdx) / (num_blks_h3b * num_blks_h2b * num_blks_h1b);
				tmp_blkIdx 		= (tmp_blkIdx) % (num_blks_h3b * num_blks_h2b * num_blks_h1b);
				blk_idx_h1b     = (tmp_blkIdx) / (num_blks_h3b * num_blks_h2b);
				tmp_blkIdx 		= (tmp_blkIdx) % (num_blks_h3b * num_blks_h2b);
				blk_idx_h2b 	= (tmp_blkIdx) / (num_blks_h3b);
				blk_idx_h3b		= blockIdx.x % (num_blks_h3b);

				// 	(3) str_blk_idx_h/p*
				str_blk_idx_h3 	= blk_idx_h3b * FUSION_SIZE_SLICE_1_H3;
				str_blk_idx_h2 	= blk_idx_h2b * FUSION_SIZE_SLICE_1_H2;
				str_blk_idx_h1 	= blk_idx_h1b * FUSION_SIZE_SLICE_1_H1;
				str_blk_idx_p6 	= blk_idx_p6b * FUSION_SIZE_SLICE_1_P6;
				str_blk_idx_p5 	= blk_idx_p5b * FUSION_SIZE_SLICE_1_P5;
				str_blk_idx_p4 	= blk_idx_p4b * FUSION_SIZE_SLICE_1_P4;

				// 	(4) rng_h/p*
				if ((base_size_h3b - (str_blk_idx_h3)) >= FUSION_SIZE_SLICE_1_H3)
					rng_h3 = FUSION_SIZE_SLICE_1_H3;
				else
					rng_h3 = base_size_h3b % FUSION_SIZE_SLICE_1_H3;
				
				if ((base_size_h2b - (str_blk_idx_h2)) >= FUSION_SIZE_SLICE_1_H2)
					rng_h2 = FUSION_SIZE_SLICE_1_H2;
				else
					rng_h2 = base_size_h2b % FUSION_SIZE_SLICE_1_H2;

				if ((base_size_h1b - (str_blk_idx_h1)) >= FUSION_SIZE_SLICE_1_H1)
					rng_h1 = FUSION_SIZE_SLICE_1_H1;
				else
					rng_h1 = base_size_h1b % FUSION_SIZE_SLICE_1_H1;
				
				if ((base_size_p6b - (str_blk_idx_p6)) >= FUSION_SIZE_SLICE_1_P6)
					rng_p6 = FUSION_SIZE_SLICE_1_P6;
				else
					rng_p6 = base_size_p6b % FUSION_SIZE_SLICE_1_P6;

				if ((base_size_p5b - (str_blk_idx_p5)) >= FUSION_SIZE_SLICE_1_P5)
					rng_p5 = FUSION_SIZE_SLICE_1_P5;
				else
					rng_p5 = base_size_p5b % FUSION_SIZE_SLICE_1_P5;

				if ((base_size_p4b - (str_blk_idx_p4)) >= FUSION_SIZE_SLICE_1_P4)
					rng_p4 = FUSION_SIZE_SLICE_1_P4;
				else
					rng_p4 = base_size_p4b % FUSION_SIZE_SLICE_1_P4;

				//  sd2_1
				if (flag_d2_1 >= 0)
				{
					// 
					double* tmp_dev_d2_t2_1 = dev_d2_t2_all + size_max_dim_d2_t2 * flag_d2_1;
					double* tmp_dev_d2_v2_1 = dev_d2_v2_all + size_max_dim_d2_v2 * flag_d2_1;

					internal_upperbound = 0;
					#pragma unroll 1
					for (int l = 0; l < base_size_p7b; l+= FUSION_SIZE_INT_UNIT)
					{
						// Part: Generalized Contraction Index (p7b)
						// internal_offset = (l + FUSION_SIZE_INT_UNIT) - size_internal;
						internal_offset = (l + FUSION_SIZE_INT_UNIT) - base_size_p7b;
						if (internal_offset > 0) internal_upperbound = internal_offset;

						// Load Input Tensor to Shared Memory: 16:16
						// # of size_internal Indices: 1
						if (idx_p6 < rng_h1 && idx_h1 < rng_h2 && threadIdx.x < FUSION_SIZE_INT_UNIT - internal_upperbound)
						for (int ll = 0; ll < rng_p4; ll++)
						{
							sm_a[threadIdx.x][threadIdx.y + ll * FUSION_SIZE_TB_1_Y] = tmp_dev_d2_t2_1[(str_blk_idx_p4 + ll + (str_blk_idx_h1 + idx_p6 + (str_blk_idx_h2 + idx_h1) * base_size_h1b) * base_size_p4b) * base_size_p7b + (threadIdx.x + l)];
						}

						// Load Input Tensor to Shared Memory
						if (idx_p6 < rng_h3 && idx_h1 < rng_p6 && threadIdx.x < FUSION_SIZE_INT_UNIT - internal_upperbound)
						for (int ll = 0; ll < rng_p5; ll++)
						{
							sm_b[threadIdx.x][threadIdx.y + ll * FUSION_SIZE_TB_1_Y] = tmp_dev_d2_v2_1[(str_blk_idx_h3 + idx_p6 + (str_blk_idx_p6 + idx_h1 + (str_blk_idx_p5 + ll) * base_size_p6b) * base_size_h3b) * base_size_p7b + (threadIdx.x + l)];
						}
						__syncthreads();

						// Cross-Product: 16
						// Part: Generalized Threads
						for (int ll = 0; ll < FUSION_SIZE_INT_UNIT - internal_upperbound; ll++)
						{   
							temp_bv[0] = sm_a[ll][idx_h1 + (idx_h2) * FUSION_SIZE_SLICE_1_H1 + 0];
							temp_bv[1] = sm_a[ll][idx_h1 + (idx_h2) * FUSION_SIZE_SLICE_1_H1 + 16];
							temp_bv[2] = sm_a[ll][idx_h1 + (idx_h2) * FUSION_SIZE_SLICE_1_H1 + 32];
							temp_bv[3] = sm_a[ll][idx_h1 + (idx_h2) * FUSION_SIZE_SLICE_1_H1 + 48];
							
							for (int xx = 0 ; xx < 4; xx++)
							{
								temp_av = sm_b[ll][idx_h3 + (idx_p6) * FUSION_SIZE_SLICE_1_H3 + (xx * 16)];

								reg_tile[0][xx] -= temp_av * temp_bv[0];
								reg_tile[1][xx] -= temp_av * temp_bv[1];
								reg_tile[2][xx] -= temp_av * temp_bv[2];
								reg_tile[3][xx] -= temp_av * temp_bv[3];
							}
						}
						__syncthreads();
					}
				}
			
				// 	sd2_2
				if (flag_d2_2 >= 0)
				{
					// 
					double* tmp_dev_d2_t2_2 = dev_d2_t2_all + size_max_dim_d2_t2 * flag_d2_2;
					double* tmp_dev_d2_v2_2 = dev_d2_v2_all + size_max_dim_d2_v2 * flag_d2_2;

					internal_upperbound = 0;
					#pragma unroll 1
					for (int l = 0; l < base_size_p7b; l+= FUSION_SIZE_INT_UNIT)
					{
						// Part: Generalized Contraction Index (p7b)
						internal_offset = (l + FUSION_SIZE_INT_UNIT) - base_size_p7b;
						if (internal_offset > 0) internal_upperbound = internal_offset;
				
						// Load Input Tensor to Shared Memory: 16:16
						// # of size_internal Indices: 1
						if (idx_p6 < rng_h2 && idx_h1 < rng_h3 && threadIdx.x < FUSION_SIZE_INT_UNIT - internal_upperbound)
						for (int ll = 0; ll < rng_p4; ll++)
						{
							sm_a[threadIdx.x][threadIdx.y + ll * FUSION_SIZE_TB_1_Y] = tmp_dev_d2_t2_2[	(str_blk_idx_p4 + ll + (str_blk_idx_h2 + idx_p6 + (str_blk_idx_h3 + idx_h1) * base_size_h2b) * base_size_p4b) * base_size_p7b + (threadIdx.x + l)];
						}
				
						// Load Input Tensor to Shared Memory
						if (idx_p6 < rng_h1 && idx_h1 < rng_p6 && threadIdx.x < FUSION_SIZE_INT_UNIT - internal_upperbound)
						for (int ll = 0; ll < rng_p5; ll++)
						{
							sm_b[threadIdx.x][threadIdx.y + ll * FUSION_SIZE_TB_1_Y] = tmp_dev_d2_v2_2[	(str_blk_idx_h1 + idx_p6 + (str_blk_idx_p6 + idx_h1 + (str_blk_idx_p5 + ll) * base_size_p6b) * base_size_h1b) * base_size_p7b + (threadIdx.x + l)];
						}
						__syncthreads();
				
						// Cross-Product: 16
						// Part: Generalized Threads
						for (int ll = 0; ll < FUSION_SIZE_INT_UNIT - internal_upperbound; ll++)
						{
							temp_bv[0] = sm_a[ll][idx_h2 + (idx_h3) * FUSION_SIZE_SLICE_1_H2 + 0];
							temp_bv[1] = sm_a[ll][idx_h2 + (idx_h3) * FUSION_SIZE_SLICE_1_H2 + 16];
							temp_bv[2] = sm_a[ll][idx_h2 + (idx_h3) * FUSION_SIZE_SLICE_1_H2 + 32];
							temp_bv[3] = sm_a[ll][idx_h2 + (idx_h3) * FUSION_SIZE_SLICE_1_H2 + 48];
				
							for (int xx = 0 ; xx < 4; xx++)
							{
								temp_av = sm_b[ll][idx_h1 + (idx_p6) * FUSION_SIZE_SLICE_1_H1 + (xx * 16)];
				
								reg_tile[0][xx] -= temp_av * temp_bv[0];
								reg_tile[1][xx] -= temp_av * temp_bv[1];
								reg_tile[2][xx] -= temp_av * temp_bv[2];
								reg_tile[3][xx] -= temp_av * temp_bv[3];
							}
						}
						__syncthreads();
					}
				}

				// 	sd2_3
				if (flag_d2_3 >= 0)
				{
					// 
					double* tmp_dev_d2_t2_3 = dev_d2_t2_all + size_max_dim_d2_t2 * flag_d2_3;
					double* tmp_dev_d2_v2_3 = dev_d2_v2_all + size_max_dim_d2_v2 * flag_d2_3;

					internal_upperbound = 0;
					#pragma unroll 1
					for (int l = 0; l < base_size_p7b; l+= FUSION_SIZE_INT_UNIT)
					{
						// Part: Generalized Contraction Index (p7b)
						internal_offset = (l + FUSION_SIZE_INT_UNIT) - base_size_p7b;
						if (internal_offset > 0) internal_upperbound = internal_offset;
				
						// Load Input Tensor to Shared Memory: 16:16
						// # of size_internal Indices: 1
						if (idx_p6 < rng_h1 && idx_h1 < rng_h3 && threadIdx.x < FUSION_SIZE_INT_UNIT - internal_upperbound)
						for (int ll = 0; ll < rng_p4; ll++)
						{
							sm_a[threadIdx.x][threadIdx.y + ll * FUSION_SIZE_TB_1_Y] = tmp_dev_d2_t2_3[	(str_blk_idx_p4 + ll + 
								(str_blk_idx_h1 + idx_p6 + 
								(str_blk_idx_h3 + idx_h1) * base_size_h1b) * base_size_p4b) * base_size_p7b + 
								(threadIdx.x + l)];
						}
				
						// Load Input Tensor to Shared Memory
						if (idx_p6 < rng_h2 && idx_h1 < rng_p6 && threadIdx.x < FUSION_SIZE_INT_UNIT - internal_upperbound)
						for (int ll = 0; ll < rng_p5; ll++)
						{
							sm_b[threadIdx.x][threadIdx.y + ll * FUSION_SIZE_TB_1_Y] = tmp_dev_d2_v2_3[	(str_blk_idx_h2 + idx_p6 + 
								(str_blk_idx_p6 + idx_h1 + 
								(str_blk_idx_p5 + ll) * base_size_p6b) * base_size_h2b) * base_size_p7b + 
								(threadIdx.x + l)];
						}
						__syncthreads();
				
						// Cross-Product: 16
						// Part: Generalized Threads
						for (int ll = 0; ll < FUSION_SIZE_INT_UNIT - internal_upperbound; ll++)
						{
							temp_bv[0] = sm_a[ll][idx_h1 + (idx_h3) * FUSION_SIZE_SLICE_1_H1 + 0];
							temp_bv[1] = sm_a[ll][idx_h1 + (idx_h3) * FUSION_SIZE_SLICE_1_H1 + 16];
							temp_bv[2] = sm_a[ll][idx_h1 + (idx_h3) * FUSION_SIZE_SLICE_1_H1 + 32];
							temp_bv[3] = sm_a[ll][idx_h1 + (idx_h3) * FUSION_SIZE_SLICE_1_H1 + 48];
				
							for (int xx = 0 ; xx < 4; xx++)
							{
								temp_av = sm_b[ll][idx_h2 + (idx_p6) * FUSION_SIZE_SLICE_1_H2 + (xx * 16)];
				
								reg_tile[0][xx] += temp_av * temp_bv[0];
								reg_tile[1][xx] += temp_av * temp_bv[1];
								reg_tile[2][xx] += temp_av * temp_bv[2];
								reg_tile[3][xx] += temp_av * temp_bv[3];
							}
						}
						__syncthreads();
					}
				}
			
				// 	sd2_4
				if (flag_d2_4 >= 0)
				{
					// 
					double* tmp_dev_d2_t2_4 = dev_d2_t2_all + size_max_dim_d2_t2 * flag_d2_4;
					double* tmp_dev_d2_v2_4 = dev_d2_v2_all + size_max_dim_d2_v2 * flag_d2_4;

					internal_upperbound = 0;
					#pragma unroll 1
					for (int l = 0; l < base_size_p7b; l+= FUSION_SIZE_INT_UNIT)
					{
						// Part: Generalized Contraction Index (p7b)
						internal_offset = (l + FUSION_SIZE_INT_UNIT) - base_size_p7b;
						if (internal_offset > 0) internal_upperbound = internal_offset;
				
						// Load Input Tensor to Shared Memory: 16:16
						// # of size_internal Indices: 1
						if (idx_p6 < rng_h1 && idx_h1 < rng_h2 && threadIdx.x < FUSION_SIZE_INT_UNIT - internal_upperbound)
						for (int ll = 0; ll < rng_p5; ll++)
						{
							sm_a[threadIdx.x][threadIdx.y + ll * FUSION_SIZE_TB_1_Y] = tmp_dev_d2_t2_4[	(str_blk_idx_p5 + ll + (str_blk_idx_h1 + idx_p6 + (str_blk_idx_h2 + idx_h1) * base_size_h1b) * base_size_p5b) * base_size_p7b + (threadIdx.x + l)];
						}
				
						// Load Input Tensor to Shared Memory
						if (idx_p6 < rng_h3 && idx_h1 < rng_p6 && threadIdx.x < FUSION_SIZE_INT_UNIT - internal_upperbound)
						for (int ll = 0; ll < rng_p4; ll++)
						{
							sm_b[threadIdx.x][threadIdx.y + ll * FUSION_SIZE_TB_1_Y] = tmp_dev_d2_v2_4[	(str_blk_idx_h3 + idx_p6 + (str_blk_idx_p6 + idx_h1 + (str_blk_idx_p4 + ll) * base_size_p6b) * base_size_h3b) * base_size_p7b + (threadIdx.x + l)];
						}
						__syncthreads();
				
						// Cross-Product: 16
						// Part: Generalized Threads
						for (int ll = 0; ll < FUSION_SIZE_INT_UNIT - internal_upperbound; ll++)
						{
							temp_bv[0] = sm_b[ll][idx_h3 + (idx_p6) * FUSION_SIZE_SLICE_1_H1 + 0];
							temp_bv[1] = sm_b[ll][idx_h3 + (idx_p6) * FUSION_SIZE_SLICE_1_H1 + 16];
							temp_bv[2] = sm_b[ll][idx_h3 + (idx_p6) * FUSION_SIZE_SLICE_1_H1 + 32];
							temp_bv[3] = sm_b[ll][idx_h3 + (idx_p6) * FUSION_SIZE_SLICE_1_H1 + 48];
				
							for (int xx = 0 ; xx < 4; xx++)
							{
								temp_av = sm_a[ll][idx_h1 + (idx_h2) * FUSION_SIZE_SLICE_1_H1 + (xx * 16)];
				
								reg_tile[0][xx] += temp_av * temp_bv[0];
								reg_tile[1][xx] += temp_av * temp_bv[1];
								reg_tile[2][xx] += temp_av * temp_bv[2];
								reg_tile[3][xx] += temp_av * temp_bv[3];
							}
						}
						__syncthreads();
					}
				}
			
				// 	sd2_5
				if (flag_d2_5 >= 0)
				{
					// 
					double* tmp_dev_d2_t2_5 = dev_d2_t2_all + size_max_dim_d2_t2 * flag_d2_5;
					double* tmp_dev_d2_v2_5 = dev_d2_v2_all + size_max_dim_d2_v2 * flag_d2_5;

					internal_upperbound = 0;
					#pragma unroll 1
					for (int l = 0; l < base_size_p7b; l+= FUSION_SIZE_INT_UNIT)
					{
						// Part: Generalized Contraction Index (p7b)
						internal_offset = (l + FUSION_SIZE_INT_UNIT) - base_size_p7b;
						if (internal_offset > 0) internal_upperbound = internal_offset;
				
						// Load Input Tensor to Shared Memory: 16:16
						// # of size_internal Indices: 1
						if (idx_p6 < rng_h2 && idx_h1 < rng_h3 && threadIdx.x < FUSION_SIZE_INT_UNIT - internal_upperbound)
						for (int ll = 0; ll < rng_p5; ll++)
						{
							sm_a[threadIdx.x][threadIdx.y + ll * FUSION_SIZE_TB_1_Y] = tmp_dev_d2_t2_5[	(str_blk_idx_p5 + ll + (str_blk_idx_h2 + idx_p6 + (str_blk_idx_h3 + idx_h1) * base_size_h2b) * base_size_p5b) * base_size_p7b + (threadIdx.x + l)];
						}
				
						// Load Input Tensor to Shared Memory
						if (idx_p6 < rng_h1 && idx_h1 < rng_p6 && threadIdx.x < FUSION_SIZE_INT_UNIT - internal_upperbound)
						for (int ll = 0; ll < rng_p4; ll++)
						{
							sm_b[threadIdx.x][threadIdx.y + ll * FUSION_SIZE_TB_1_Y] = tmp_dev_d2_v2_5[	(str_blk_idx_h1 + idx_p6 + (str_blk_idx_p6 + idx_h1 + (str_blk_idx_p4 + ll) * base_size_p6b) * base_size_h1b) * base_size_p7b + (threadIdx.x + l)];
						}
						__syncthreads();
				
						// Cross-Product: 16
						// Part: Generalized Threads
						for (int ll = 0; ll < FUSION_SIZE_INT_UNIT - internal_upperbound; ll++)
						{
							temp_bv[0] = sm_b[ll][idx_h1 + (idx_p6) * FUSION_SIZE_SLICE_1_H1 + 0];
							temp_bv[1] = sm_b[ll][idx_h1 + (idx_p6) * FUSION_SIZE_SLICE_1_H1 + 16];
							temp_bv[2] = sm_b[ll][idx_h1 + (idx_p6) * FUSION_SIZE_SLICE_1_H1 + 32];
							temp_bv[3] = sm_b[ll][idx_h1 + (idx_p6) * FUSION_SIZE_SLICE_1_H1 + 48];
				
							for (int xx = 0 ; xx < 4; xx++)
							{
								temp_av = sm_a[ll][idx_h2 + (idx_h3) * FUSION_SIZE_SLICE_1_H2 + (xx * 16)];
				
								reg_tile[0][xx] += temp_av * temp_bv[0];
								reg_tile[1][xx] += temp_av * temp_bv[1];
								reg_tile[2][xx] += temp_av * temp_bv[2];
								reg_tile[3][xx] += temp_av * temp_bv[3];
							}
						}
						__syncthreads();
					}
				}
			
				// 	sd2_6
				if (flag_d2_6 >= 0)
				{
					// 
					double* tmp_dev_d2_t2_6 = dev_d2_t2_all + size_max_dim_d2_t2 * flag_d2_6;
					double* tmp_dev_d2_v2_6 = dev_d2_v2_all + size_max_dim_d2_v2 * flag_d2_6;

					// if (blockIdx.x == 0 && threadIdx.x == 0 && threadIdx.y == 0)
					// {
					// 	printf ("d2_t2_6[0] = %.15f\n", tmp_dev_d2_t2_6[0]);
					// 	printf ("d2_t2_6[1] = %.15f\n", tmp_dev_d2_t2_6[1]);
					// }

					internal_upperbound = 0;
					#pragma unroll 1
					for (int l = 0; l < base_size_p7b; l+= FUSION_SIZE_INT_UNIT)
					{
						// Part: Generalized Contraction Index (p7b)
						internal_offset = (l + FUSION_SIZE_INT_UNIT) - base_size_p7b;
						if (internal_offset > 0) internal_upperbound = internal_offset;
				
						// Load Input Tensor to Shared Memory: 16:16
						// # of size_internal Indices: 1
						if (idx_p6 < rng_h1 && idx_h1 < rng_h3 && threadIdx.x < FUSION_SIZE_INT_UNIT - internal_upperbound)
						for (int ll = 0; ll < rng_p5; ll++)
						{
							sm_a[threadIdx.x][threadIdx.y + ll * FUSION_SIZE_TB_1_Y] = tmp_dev_d2_t2_6[	(blk_idx_p5b * FUSION_SIZE_SLICE_1_P6 + ll + (str_blk_idx_h1 + idx_p6 + (str_blk_idx_h3 + idx_h1) * base_size_h1b) * base_size_p5b) * base_size_p7b + (threadIdx.x + l)];
						}
				
						// Load Input Tensor to Shared Memory
						if (idx_p6 < rng_h2 && idx_h1 < rng_p6 && threadIdx.x < FUSION_SIZE_INT_UNIT - internal_upperbound)
						for (int ll = 0; ll < rng_p4; ll++)
						{
							sm_b[threadIdx.x][threadIdx.y + ll * FUSION_SIZE_TB_1_Y] = tmp_dev_d2_v2_6[	(str_blk_idx_h2 + idx_p6 + (str_blk_idx_p6 + idx_h1 + (str_blk_idx_p4 + ll) * base_size_p6b) * base_size_h2b) * base_size_p7b + (threadIdx.x + l)];
						}
						__syncthreads();
				
						// Cross-Product: 16
						// Part: Generalized Threads
						for (int ll = 0; ll < FUSION_SIZE_INT_UNIT - internal_upperbound; ll++)
						{
							temp_bv[0] = sm_b[ll][idx_h2 + (idx_p6) * FUSION_SIZE_SLICE_1_H2 + 0];
							temp_bv[1] = sm_b[ll][idx_h2 + (idx_p6) * FUSION_SIZE_SLICE_1_H2 + 16];
							temp_bv[2] = sm_b[ll][idx_h2 + (idx_p6) * FUSION_SIZE_SLICE_1_H2 + 32];
							temp_bv[3] = sm_b[ll][idx_h2 + (idx_p6) * FUSION_SIZE_SLICE_1_H2 + 48];
				
							for (int xx = 0; xx < 4; xx++)	// 4 -> rng_p4: Local Transactions...
							{
								temp_av = sm_a[ll][idx_h1 + (idx_h3) * FUSION_SIZE_SLICE_1_H1 + (xx * 16)];
				
								reg_tile[0][xx] -= temp_av * temp_bv[0];
								reg_tile[1][xx] -= temp_av * temp_bv[1];
								reg_tile[2][xx] -= temp_av * temp_bv[2];
								reg_tile[3][xx] -= temp_av * temp_bv[3];
							}
						}
						__syncthreads();
					}
				}
			}
		
		}

		//  singles (s1)
		{	
			// 	flags
			int flag_s1_1 = const_list_s1_flags_offset[0 + iter_ia6 * NUM_S1_EQUATIONS];
			int flag_s1_2 = const_list_s1_flags_offset[1 + iter_ia6 * NUM_S1_EQUATIONS];
			int flag_s1_3 = const_list_s1_flags_offset[2 + iter_ia6 * NUM_S1_EQUATIONS];
			int flag_s1_4 = const_list_s1_flags_offset[3 + iter_ia6 * NUM_S1_EQUATIONS];
			int flag_s1_5 = const_list_s1_flags_offset[4 + iter_ia6 * NUM_S1_EQUATIONS];
			int flag_s1_6 = const_list_s1_flags_offset[5 + iter_ia6 * NUM_S1_EQUATIONS];
			int flag_s1_7 = const_list_s1_flags_offset[6 + iter_ia6 * NUM_S1_EQUATIONS];
			int flag_s1_8 = const_list_s1_flags_offset[7 + iter_ia6 * NUM_S1_EQUATIONS];
			int flag_s1_9 = const_list_s1_flags_offset[8 + iter_ia6 * NUM_S1_EQUATIONS];

			// if (blockIdx.x == 0 && threadIdx.x == 0 && threadIdx.y == 0)
			// {
			// 	printf ("[Device][s1] ia6=%d, flag_s1_(1, 2, 3, 4, 5, 6, 7, 8, 9) = (%2d,%2d,%2d,%2d,%2d,%2d,%2d,%2d,%2d)\n", iter_ia6, flag_s1_1, flag_s1_2, flag_s1_3, flag_s1_4, flag_s1_5, flag_s1_6, flag_s1_7, flag_s1_8, flag_s1_9);
			// }

			// 	problem-sizes
			// int local_s1_size_idx_h1b = const_list_s1_problem_size[0 + iter_ia6 * NUM_S1_INDEX];
			// int local_s1_size_idx_h2b = const_list_s1_problem_size[1 + iter_ia6 * NUM_S1_INDEX];
			// int local_s1_size_idx_h3b = const_list_s1_problem_size[2 + iter_ia6 * NUM_S1_INDEX];
			// int local_s1_size_idx_p4b = const_list_s1_problem_size[3 + iter_ia6 * NUM_S1_INDEX];
			// int local_s1_size_idx_p5b = const_list_s1_problem_size[4 + iter_ia6 * NUM_S1_INDEX];
			// int local_s1_size_idx_p6b = const_list_s1_problem_size[5 + iter_ia6 * NUM_S1_INDEX];
			base_size_h1b = const_list_s1_problem_size[0 + iter_ia6 * NUM_S1_INDEX];
			base_size_h2b = const_list_s1_problem_size[1 + iter_ia6 * NUM_S1_INDEX];
			base_size_h3b = const_list_s1_problem_size[2 + iter_ia6 * NUM_S1_INDEX];
			base_size_p4b = const_list_s1_problem_size[3 + iter_ia6 * NUM_S1_INDEX];
			base_size_p5b = const_list_s1_problem_size[4 + iter_ia6 * NUM_S1_INDEX];
			base_size_p6b = const_list_s1_problem_size[5 + iter_ia6 * NUM_S1_INDEX];

			// 
			// if (blockIdx.x == 0 && threadIdx.x == 0 && threadIdx.y == 0)
			// {
			// 	printf ("[s1][ia6 = %2d] h1,h2,h3,p4,p5,p6 = %2d,%2d,%2d,%2d,%2d,%2d\n", iter_ia6, base_size_h1b, base_size_h2b, base_size_h3b, base_size_p4b, base_size_p5b, base_size_p6b);
			// }

			//	otheres according to the above problem-sizes
			//	(1) num_blks_h/p*b
			num_blks_h1b = CEIL(base_size_h1b, FUSION_SIZE_SLICE_1_H1);
			num_blks_h2b = CEIL(base_size_h2b, FUSION_SIZE_SLICE_1_H2);
			num_blks_h3b = CEIL(base_size_h3b, FUSION_SIZE_SLICE_1_H3);
			num_blks_p4b = CEIL(base_size_p4b, FUSION_SIZE_SLICE_1_P4);
			num_blks_p5b = CEIL(base_size_p5b, FUSION_SIZE_SLICE_1_P5);
			num_blks_p6b = CEIL(base_size_p6b, FUSION_SIZE_SLICE_1_P6);

			// 	(2) blk_idx_h/p*b
			blk_idx_p4b     = blockIdx.x / (num_blks_h3b * num_blks_h2b * num_blks_h1b * num_blks_p6b * num_blks_p5b);
			tmp_blkIdx      = blockIdx.x % (num_blks_h3b * num_blks_h2b * num_blks_h1b * num_blks_p6b * num_blks_p5b);
			blk_idx_p5b     = (tmp_blkIdx) / (num_blks_h3b * num_blks_h2b * num_blks_h1b * num_blks_p6b);
			tmp_blkIdx  	= (tmp_blkIdx) % (num_blks_h3b * num_blks_h2b * num_blks_h1b * num_blks_p6b);
			blk_idx_p6b     = (tmp_blkIdx) / (num_blks_h3b * num_blks_h2b * num_blks_h1b);
			tmp_blkIdx 		= (tmp_blkIdx) % (num_blks_h3b * num_blks_h2b * num_blks_h1b);
			blk_idx_h1b     = (tmp_blkIdx) / (num_blks_h3b * num_blks_h2b);
			tmp_blkIdx 		= (tmp_blkIdx) % (num_blks_h3b * num_blks_h2b);
			blk_idx_h2b 	= (tmp_blkIdx) / (num_blks_h3b);
			blk_idx_h3b		= blockIdx.x % (num_blks_h3b);

			// 	(3) str_blk_idx_h/p*
			str_blk_idx_h3 	= blk_idx_h3b * FUSION_SIZE_SLICE_1_H3;
			str_blk_idx_h2 	= blk_idx_h2b * FUSION_SIZE_SLICE_1_H2;
			str_blk_idx_h1 	= blk_idx_h1b * FUSION_SIZE_SLICE_1_H1;
			str_blk_idx_p6 	= blk_idx_p6b * FUSION_SIZE_SLICE_1_P6;
			str_blk_idx_p5 	= blk_idx_p5b * FUSION_SIZE_SLICE_1_P5;
			str_blk_idx_p4 	= blk_idx_p4b * FUSION_SIZE_SLICE_1_P4;

			// 	(4) rng_h/p*
			if ((base_size_h3b - (str_blk_idx_h3)) >= FUSION_SIZE_SLICE_1_H3)
				rng_h3 = FUSION_SIZE_SLICE_1_H3;
			else
				rng_h3 = base_size_h3b % FUSION_SIZE_SLICE_1_H3;
			
			if ((base_size_h2b - (str_blk_idx_h2)) >= FUSION_SIZE_SLICE_1_H2)
				rng_h2 = FUSION_SIZE_SLICE_1_H2;
			else
				rng_h2 = base_size_h2b % FUSION_SIZE_SLICE_1_H2;

			if ((base_size_h1b - (str_blk_idx_h1)) >= FUSION_SIZE_SLICE_1_H1)
				rng_h1 = FUSION_SIZE_SLICE_1_H1;
			else
				rng_h1 = base_size_h1b % FUSION_SIZE_SLICE_1_H1;
			
			if ((base_size_p6b - (str_blk_idx_p6)) >= FUSION_SIZE_SLICE_1_P6)
				rng_p6 = FUSION_SIZE_SLICE_1_P6;
			else
				rng_p6 = base_size_p6b % FUSION_SIZE_SLICE_1_P6;

			if ((base_size_p5b - (str_blk_idx_p5)) >= FUSION_SIZE_SLICE_1_P5)
				rng_p5 = FUSION_SIZE_SLICE_1_P5;
			else
				rng_p5 = base_size_p5b % FUSION_SIZE_SLICE_1_P5;

			if ((base_size_p4b - (str_blk_idx_p4)) >= FUSION_SIZE_SLICE_1_P4)
				rng_p4 = FUSION_SIZE_SLICE_1_P4;
			else
				rng_p4 = base_size_p4b % FUSION_SIZE_SLICE_1_P4;

			//                                        "x"         "x"
			//  >> s1_1:   t3[h3,h2,h1,p6,p5,p4] -= t2[p4,h1] * v2[h3,h2,p6,p5]
			//
			if (flag_s1_1 >= 0)	// these if-conditions make 100 ms..
			{
				//
				double* tmp_dev_s1_t2_1 = dev_s1_t2_all + size_max_dim_s1_t2 * flag_s1_1;
				double* tmp_dev_s1_v2_1 = dev_s1_v2_all + size_max_dim_s1_v2 * flag_s1_1;

				if (idx_h3 < rng_p4 && idx_h2 < rng_h1 && idx_p6 == 0 && idx_h1 == 0)
				sm_a[0][idx_h3 + (idx_h2) * FUSION_SIZE_SLICE_1_P4] = tmp_dev_s1_t2_1[str_blk_idx_p4 + idx_h3 + (str_blk_idx_h1 + idx_h2) * base_size_p4b];

				if (idx_h3 < rng_h3 && idx_h2 < rng_h2 && idx_p6 < rng_p6 && idx_h1 < rng_p5)
				sm_b[idx_h1][idx_h3 + (idx_h2 + (idx_p6) * 4) * 4] = tmp_dev_s1_v2_1[blk_idx_h3b * 4 + idx_h3 + (blk_idx_h2b * 4 + idx_h2 + (blk_idx_p6b * 4 + idx_p6 + (blk_idx_p5b * 4 + idx_h1) * base_size_p6b) * base_size_h2b) * base_size_h3b];
				__syncthreads();

				//  "p4"
				temp_av = sm_a[0][0 + (idx_h1) * 4];
				
				//  "p5"
				temp_bv[0] = sm_b[0][idx_h3 + (idx_h2 + (idx_p6) * 4) * 4];
				temp_bv[1] = sm_b[1][idx_h3 + (idx_h2 + (idx_p6) * 4) * 4];
				temp_bv[2] = sm_b[2][idx_h3 + (idx_h2 + (idx_p6) * 4) * 4];
				temp_bv[3] = sm_b[3][idx_h3 + (idx_h2 + (idx_p6) * 4) * 4];

				//  "p4 x p5"
				reg_singles[0][0] += temp_av * temp_bv[0];// * reg_singles[0][0];
				reg_singles[0][1] += temp_av * temp_bv[1];// * reg_singles[0][1];
				reg_singles[0][2] += temp_av * temp_bv[2];// * reg_singles[0][2];
				reg_singles[0][3] += temp_av * temp_bv[3];// * reg_singles[0][3];

				temp_av = sm_a[0][1 + (idx_h1) * 4];
				
				reg_singles[1][0] += temp_av * temp_bv[0];// * reg_singles[1][0];
				reg_singles[1][1] += temp_av * temp_bv[1];// * reg_singles[1][1];
				reg_singles[1][2] += temp_av * temp_bv[2];// * reg_singles[1][2];
				reg_singles[1][3] += temp_av * temp_bv[3];// * reg_singles[1][3];
				
				temp_av = sm_a[0][2 + (idx_h1) * 4];

				reg_singles[2][0] += temp_av * temp_bv[0];// * reg_singles[2][0];
				reg_singles[2][1] += temp_av * temp_bv[1];// * reg_singles[2][1];
				reg_singles[2][2] += temp_av * temp_bv[2];// * reg_singles[2][2];
				reg_singles[2][3] += temp_av * temp_bv[3];// * reg_singles[2][3];

				temp_av = sm_a[0][3 + (idx_h1) * 4];

				reg_singles[3][0] += temp_av * temp_bv[0];// * reg_singles[3][0];
				reg_singles[3][1] += temp_av * temp_bv[1];// * reg_singles[3][1];
				reg_singles[3][2] += temp_av * temp_bv[2];// * reg_singles[3][2];
				reg_singles[3][3] += temp_av * temp_bv[3];// * reg_singles[3][3];
				__syncthreads();
			}

			//                                        "x1,x2"     "x1,x2,x3,y1"
			//  >> s1_2:   t3[h3,h2,h1,p6,p5,p4] -= t2[p4,h2] * v2[h3,h1,p6,p5] (h3,h2,p6), (h1)
			//
			if (flag_s1_2 >= 0)	// these if-conditions make 100 ms..
			{
				// 
				double* tmp_dev_s1_t2_2 = dev_s1_t2_all + size_max_dim_s1_t2 * flag_s1_2;
				double* tmp_dev_s1_v2_2 = dev_s1_v2_all + size_max_dim_s1_v2 * flag_s1_2;

				if (idx_h3 < rng_p4 && idx_h2 < rng_h2 && idx_p6 == 0 && idx_h1 == 0)
				sm_a[0][idx_h3 + (idx_h2) * FUSION_SIZE_SLICE_1_P4] = tmp_dev_s1_t2_2[str_blk_idx_p4 + idx_h3 + (str_blk_idx_h2 + idx_h2) * base_size_p4b];
				
				if (idx_h3 < rng_h3 && idx_h2 < rng_h1 && idx_p6 < rng_p6 && idx_h1 < rng_p5)
				sm_b[idx_h1][idx_h3 + (idx_h2 + (idx_p6) * FUSION_SIZE_SLICE_1_H1) * FUSION_SIZE_SLICE_1_H3] = tmp_dev_s1_v2_2[str_blk_idx_h3 + idx_h3 + (str_blk_idx_h1 + idx_h2 + (str_blk_idx_p6 + idx_p6 + (str_blk_idx_p5 + idx_h1) * base_size_p6b) * base_size_h1b) * base_size_h3b];
				__syncthreads();

				//  "p4"
				temp_av = sm_a[0][0 + (idx_h2) * 4];
				
				//  "p5"
				temp_bv[0] = sm_b[0][idx_h3 + (idx_h1 + (idx_p6) * 4) * 4];
				temp_bv[1] = sm_b[1][idx_h3 + (idx_h1 + (idx_p6) * 4) * 4];
				temp_bv[2] = sm_b[2][idx_h3 + (idx_h1 + (idx_p6) * 4) * 4];
				temp_bv[3] = sm_b[3][idx_h3 + (idx_h1 + (idx_p6) * 4) * 4];

				//  "p4 x p5"
				reg_singles[0][0] -= temp_av * temp_bv[0];
				reg_singles[0][1] -= temp_av * temp_bv[1];
				reg_singles[0][2] -= temp_av * temp_bv[2];
				reg_singles[0][3] -= temp_av * temp_bv[3];

				temp_av = sm_a[0][1 + (idx_h2) * 4];

				reg_singles[1][0] -= temp_av * temp_bv[0];
				reg_singles[1][1] -= temp_av * temp_bv[1];
				reg_singles[1][2] -= temp_av * temp_bv[2];
				reg_singles[1][3] -= temp_av * temp_bv[3];

				temp_av = sm_a[0][2 + (idx_h2) * 4];

				reg_singles[2][0] -= temp_av * temp_bv[0];
				reg_singles[2][1] -= temp_av * temp_bv[1];
				reg_singles[2][2] -= temp_av * temp_bv[2];
				reg_singles[2][3] -= temp_av * temp_bv[3];

				temp_av = sm_a[0][3 + (idx_h2) * 4];

				reg_singles[3][0] -= temp_av * temp_bv[0];
				reg_singles[3][1] -= temp_av * temp_bv[1];
				reg_singles[3][2] -= temp_av * temp_bv[2];
				reg_singles[3][3] -= temp_av * temp_bv[3];
				__syncthreads();
			}

			//
			//  >> s1_3:   t3[h3,h2,h1,p6,p5,p4] -= t2[p4,h1] * v2[h3,h2,p6,p5] >> t3[h3,h2,h1,p6,p5,p4] += t2[p4,h3] * v2[h2,h1,p6,p5]
			//
			if (flag_s1_3 >= 0)	// these if-conditions make 100 ms..
			{
				// 
				double* tmp_dev_s1_t2_3 = dev_s1_t2_all + size_max_dim_s1_t2 * flag_s1_3;
				double* tmp_dev_s1_v2_3 = dev_s1_v2_all + size_max_dim_s1_v2 * flag_s1_3;

				if (idx_h3 < rng_p4 && idx_h2 < rng_h3 && idx_p6 == 0 && idx_h1 == 0)
				sm_a[0][idx_h3 + (idx_h2) * FUSION_SIZE_SLICE_1_P4] = tmp_dev_s1_t2_3[str_blk_idx_p4 + idx_h3 + (str_blk_idx_h3 + idx_h2) * base_size_p4b];

				if (idx_h3 < rng_h2 && idx_h2 < rng_h1 && idx_p6 < rng_p6 && idx_h1 < rng_p5)
				sm_b[idx_h1][idx_h3 + (idx_h2 + (idx_p6) * 4) * 4] = tmp_dev_s1_v2_3[blk_idx_h2b * 4 + idx_h3 + (blk_idx_h1b * 4 + idx_h2 + (blk_idx_p6b * 4 + idx_p6 + (blk_idx_p5b * 4 + idx_h1) * base_size_p6b) * base_size_h1b) * base_size_h2b];
				__syncthreads();

				//  "p4"
				temp_av = sm_a[0][0 + (idx_h3) * 4];
				
				//  "p5"
				temp_bv[0] = sm_b[0][idx_h2 + (idx_h1 + (idx_p6) * 4) * 4];
				temp_bv[1] = sm_b[1][idx_h2 + (idx_h1 + (idx_p6) * 4) * 4];
				temp_bv[2] = sm_b[2][idx_h2 + (idx_h1 + (idx_p6) * 4) * 4];
				temp_bv[3] = sm_b[3][idx_h2 + (idx_h1 + (idx_p6) * 4) * 4];

				//  "p4 x p5"
				reg_singles[0][0] += temp_av * temp_bv[0];
				reg_singles[0][1] += temp_av * temp_bv[1];
				reg_singles[0][2] += temp_av * temp_bv[2];
				reg_singles[0][3] += temp_av * temp_bv[3];

				temp_av = sm_a[0][1 + (idx_h3) * 4];

				reg_singles[1][0] += temp_av * temp_bv[0];
				reg_singles[1][1] += temp_av * temp_bv[1];
				reg_singles[1][2] += temp_av * temp_bv[2];
				reg_singles[1][3] += temp_av * temp_bv[3];

				temp_av = sm_a[0][2 + (idx_h3) * 4];

				reg_singles[2][0] += temp_av * temp_bv[0];
				reg_singles[2][1] += temp_av * temp_bv[1];
				reg_singles[2][2] += temp_av * temp_bv[2];
				reg_singles[2][3] += temp_av * temp_bv[3];

				temp_av = sm_a[0][3 + (idx_h3) * 4];

				reg_singles[3][0] += temp_av * temp_bv[0];
				reg_singles[3][1] += temp_av * temp_bv[1];
				reg_singles[3][2] += temp_av * temp_bv[2];
				reg_singles[3][3] += temp_av * temp_bv[3];
				__syncthreads();
			}
		
			//
			//  >> s1_4:   t3[h3,h2,h1,p6,p5,p4] -= t2[p5,h1] * v2[h3,h2,p6,p4] (h3,h2,p6), (h1)
			//
			if (flag_s1_4 >= 0)	// these if-conditions make 100 ms..
			{
				double* tmp_dev_s1_t2_4 = dev_s1_t2_all + size_max_dim_s1_t2 * flag_s1_4;
				double* tmp_dev_s1_v2_4 = dev_s1_v2_all + size_max_dim_s1_v2 * flag_s1_4;

				if (idx_h3 < rng_p5 && idx_h2 < rng_h1 && idx_p6 == 0 && idx_h1 == 0)
				sm_a[0][idx_h3 + (idx_h2) * FUSION_SIZE_SLICE_1_P5] = tmp_dev_s1_t2_4[str_blk_idx_p5 + idx_h3 + (str_blk_idx_h1 + idx_h2) * base_size_p5b];
				
				if (idx_h3 < rng_h3 && idx_h2 < rng_h2 && idx_p6 < rng_p6 && idx_h1 < rng_p4)
				sm_b[idx_h1][idx_h3 + (idx_h2 + (idx_p6) * 4) * 4] = tmp_dev_s1_v2_4[str_blk_idx_h3 + idx_h3 + (str_blk_idx_h2 + idx_h2 + (str_blk_idx_p6 + idx_p6 + (str_blk_idx_p4 + idx_h1) * base_size_p6b) * base_size_h2b) * base_size_h3b];
				__syncthreads();

				//  "p5"
				temp_av = sm_a[0][0 + (idx_h1) * 4];
				
				//  "p4"
				temp_bv[0] = sm_b[0][idx_h3 + (idx_h2 + (idx_p6) * 4) * 4];
				temp_bv[1] = sm_b[1][idx_h3 + (idx_h2 + (idx_p6) * 4) * 4];
				temp_bv[2] = sm_b[2][idx_h3 + (idx_h2 + (idx_p6) * 4) * 4];
				temp_bv[3] = sm_b[3][idx_h3 + (idx_h2 + (idx_p6) * 4) * 4];

				//  "p4 x p5"
				reg_singles[0][0] -= temp_av * temp_bv[0];
				reg_singles[1][0] -= temp_av * temp_bv[1];
				reg_singles[2][0] -= temp_av * temp_bv[2];
				reg_singles[3][0] -= temp_av * temp_bv[3];

				temp_av = sm_a[0][1 + (idx_h1) * 4];

				reg_singles[0][1] -= temp_av * temp_bv[0];
				reg_singles[1][1] -= temp_av * temp_bv[1];
				reg_singles[2][1] -= temp_av * temp_bv[2];
				reg_singles[3][1] -= temp_av * temp_bv[3];

				temp_av = sm_a[0][2 + (idx_h1) * 4];

				reg_singles[0][2] -= temp_av * temp_bv[0];
				reg_singles[1][2] -= temp_av * temp_bv[1];
				reg_singles[2][2] -= temp_av * temp_bv[2];
				reg_singles[3][2] -= temp_av * temp_bv[3];

				temp_av = sm_a[0][3 + (idx_h1) * 4];

				reg_singles[0][3] -= temp_av * temp_bv[0];
				reg_singles[1][3] -= temp_av * temp_bv[1];
				reg_singles[2][3] -= temp_av * temp_bv[2];
				reg_singles[3][3] -= temp_av * temp_bv[3];
				__syncthreads();
			}

			//
			//  >> s1_5:   t3[h3,h2,h1,p6,p5,p4] -= t2[p5,h2] * v2[h3,h1,p6,p4] (h3,h2,p6), (h1)
			//
			if (flag_s1_5 >= 0)	// these if-conditions make 100 ms..
			{
				// 
				double* tmp_dev_s1_t2_5 = dev_s1_t2_all + size_max_dim_s1_t2 * flag_s1_5;
				double* tmp_dev_s1_v2_5 = dev_s1_v2_all + size_max_dim_s1_v2 * flag_s1_5;

				if (idx_h3 < rng_p5 && idx_h2 < rng_h2 && idx_p6 == 0 && idx_h1 == 0)
				sm_a[0][idx_h3 + (idx_h2) * FUSION_SIZE_SLICE_1_P5] = tmp_dev_s1_t2_5[str_blk_idx_p5 + idx_h3 + (str_blk_idx_h2 + idx_h2) * base_size_p5b];

				if (idx_h3 < rng_h3 && idx_h2 < rng_h1 && idx_p6 < rng_p6 && idx_h1 < rng_p4)
				sm_b[idx_h1][idx_h3 + (idx_h2 + (idx_p6) * FUSION_SIZE_SLICE_1_H1) * FUSION_SIZE_SLICE_1_H3] = tmp_dev_s1_v2_5[str_blk_idx_h3 + idx_h3 + (str_blk_idx_h1 + idx_h2 + (str_blk_idx_p6 + idx_p6 + (str_blk_idx_p4 + idx_h1) * base_size_p6b) * base_size_h1b) * base_size_h3b];
				__syncthreads();

				//  "p5"
				temp_av = sm_a[0][0 + (idx_h2) * 4];
				
				//  "p4"
				temp_bv[0] = sm_b[0][idx_h3 + (idx_h1 + (idx_p6) * 4) * 4];
				temp_bv[1] = sm_b[1][idx_h3 + (idx_h1 + (idx_p6) * 4) * 4];
				temp_bv[2] = sm_b[2][idx_h3 + (idx_h1 + (idx_p6) * 4) * 4];
				temp_bv[3] = sm_b[3][idx_h3 + (idx_h1 + (idx_p6) * 4) * 4];

				//  "p4 x p5"
				reg_singles[0][0] += temp_av * temp_bv[0];
				reg_singles[1][0] += temp_av * temp_bv[1];
				reg_singles[2][0] += temp_av * temp_bv[2];
				reg_singles[3][0] += temp_av * temp_bv[3];
				
				temp_av = sm_a[0][1 + (idx_h2) * 4];

				reg_singles[0][1] += temp_av * temp_bv[0];
				reg_singles[1][1] += temp_av * temp_bv[1];
				reg_singles[2][1] += temp_av * temp_bv[2];
				reg_singles[3][1] += temp_av * temp_bv[3];

				temp_av = sm_a[0][2 + (idx_h2) * 4];

				reg_singles[0][2] += temp_av * temp_bv[0];
				reg_singles[1][2] += temp_av * temp_bv[1];
				reg_singles[2][2] += temp_av * temp_bv[2];
				reg_singles[3][2] += temp_av * temp_bv[3];

				temp_av = sm_a[0][3 + (idx_h2) * 4];

				reg_singles[0][3] += temp_av * temp_bv[0];
				reg_singles[1][3] += temp_av * temp_bv[1];
				reg_singles[2][3] += temp_av * temp_bv[2];
				reg_singles[3][3] += temp_av * temp_bv[3];
				__syncthreads();
			}
			
			//
			//  >> s1_6:   t3[h3,h2,h1,p6,p5,p4] -= t2[p5,h3] * v2[h2,h1,p6,p4] (h3,h2,p6), (h1)
			//
			if (flag_s1_6 >= 0)	// these if-conditions make 100 ms..
			{
				// 
				double* tmp_dev_s1_t2_6 = dev_s1_t2_all + size_max_dim_s1_t2 * flag_s1_6;
				double* tmp_dev_s1_v2_6 = dev_s1_v2_all + size_max_dim_s1_v2 * flag_s1_6;

				if (idx_h3 < rng_p5 && idx_h2 < rng_h3 && idx_p6 == 0 && idx_h1 == 0)
				sm_a[0][idx_h3 + (idx_h2) * FUSION_SIZE_SLICE_1_P5] = tmp_dev_s1_t2_6[str_blk_idx_p5 + idx_h3 + (str_blk_idx_h3 + idx_h2) * base_size_p5b];

				if (idx_h3 < rng_h2 && idx_h2 < rng_h1 && idx_p6 < rng_p6 && idx_h1 < rng_p4)
				sm_b[idx_h1][idx_h3 + (idx_h2 + (idx_p6) * FUSION_SIZE_SLICE_1_H1) * FUSION_SIZE_SLICE_1_H2] = tmp_dev_s1_v2_6[str_blk_idx_h2 + idx_h3 + (str_blk_idx_h1 + idx_h2 + (str_blk_idx_p6 + idx_p6 + (str_blk_idx_p4 + idx_h1) * base_size_p6b) * base_size_h1b) * base_size_h2b];
				__syncthreads();

				//  "p5"
				temp_av = sm_a[0][0 + (idx_h3) * FUSION_SIZE_SLICE_1_P5];
				
				//  "p4"
				temp_bv[0] = sm_b[0][idx_h2 + (idx_h1 + (idx_p6) * 4) * 4];
				temp_bv[1] = sm_b[1][idx_h2 + (idx_h1 + (idx_p6) * 4) * 4];
				temp_bv[2] = sm_b[2][idx_h2 + (idx_h1 + (idx_p6) * 4) * 4];
				temp_bv[3] = sm_b[3][idx_h2 + (idx_h1 + (idx_p6) * 4) * 4];

				//  "p4 x p5"
				reg_singles[0][0] -= temp_av * temp_bv[0];
				reg_singles[1][0] -= temp_av * temp_bv[1];
				reg_singles[2][0] -= temp_av * temp_bv[2];
				reg_singles[3][0] -= temp_av * temp_bv[3];
				
				temp_av = sm_a[0][1 + (idx_h3) * FUSION_SIZE_SLICE_1_P5];

				reg_singles[0][1] -= temp_av * temp_bv[0];
				reg_singles[1][1] -= temp_av * temp_bv[1];
				reg_singles[2][1] -= temp_av * temp_bv[2];
				reg_singles[3][1] -= temp_av * temp_bv[3];
				
				temp_av = sm_a[0][2 + (idx_h3) * FUSION_SIZE_SLICE_1_P5];

				reg_singles[0][2] -= temp_av * temp_bv[0];
				reg_singles[1][2] -= temp_av * temp_bv[1];
				reg_singles[2][2] -= temp_av * temp_bv[2];
				reg_singles[3][2] -= temp_av * temp_bv[3];

				temp_av = sm_a[0][3 + (idx_h3) * FUSION_SIZE_SLICE_1_P5];

				reg_singles[0][3] -= temp_av * temp_bv[0];
				reg_singles[1][3] -= temp_av * temp_bv[1];
				reg_singles[2][3] -= temp_av * temp_bv[2];
				reg_singles[3][3] -= temp_av * temp_bv[3];
				__syncthreads();
			}
			
			//
			//  >> s1_7:   t3[h3,h2,h1,p6,p5,p4] -= t2[p6,h1] * v2[h3,h2,p5,p4] (h3,h2,p6), (h1)
			//
			if (flag_s1_7 >= 0)	// these if-conditions make 100 ms..
			{
				//
				double* tmp_dev_s1_t2_7 = dev_s1_t2_all + size_max_dim_s1_t2 * flag_s1_7;
				double* tmp_dev_s1_v2_7 = dev_s1_v2_all + size_max_dim_s1_v2 * flag_s1_7;

				if (idx_h3 < rng_p6 && idx_h2 < rng_h1 && idx_p6 == 0 && idx_h1 == 0)
				sm_a[0][idx_h3 + (idx_h2) * FUSION_SIZE_SLICE_1_P6] = tmp_dev_s1_t2_7[str_blk_idx_p6 + idx_h3 + (str_blk_idx_h1 + idx_h2) * base_size_p6b];
				
				if (idx_h3 < rng_h3 && idx_h2 < rng_h2 && idx_p6 < rng_p5 && idx_h1 < rng_p4)
				sm_b[idx_h1][idx_h3 + (idx_h2 + (idx_p6) * FUSION_SIZE_SLICE_1_H2) * FUSION_SIZE_SLICE_1_H3] = tmp_dev_s1_v2_7[str_blk_idx_h3 + idx_h3 + (str_blk_idx_h2 + idx_h2 + (str_blk_idx_p5 + idx_p6 + (str_blk_idx_p4 + idx_h1) * base_size_p5b) * base_size_h2b) * base_size_h3b];
				__syncthreads();

				//  "p4" x "p5"
				reg_singles[0][0] += sm_a[0][idx_p6 + (idx_h1) * FUSION_SIZE_SLICE_1_P6] * sm_b[0][idx_h3 + (idx_h2 + (0) * FUSION_SIZE_SLICE_1_H2) * FUSION_SIZE_SLICE_1_H3];
				reg_singles[0][1] += sm_a[0][idx_p6 + (idx_h1) * FUSION_SIZE_SLICE_1_P6] * sm_b[0][idx_h3 + (idx_h2 + (1) * FUSION_SIZE_SLICE_1_H2) * FUSION_SIZE_SLICE_1_H3];
				reg_singles[0][2] += sm_a[0][idx_p6 + (idx_h1) * FUSION_SIZE_SLICE_1_P6] * sm_b[0][idx_h3 + (idx_h2 + (2) * FUSION_SIZE_SLICE_1_H2) * FUSION_SIZE_SLICE_1_H3];
				reg_singles[0][3] += sm_a[0][idx_p6 + (idx_h1) * FUSION_SIZE_SLICE_1_P6] * sm_b[0][idx_h3 + (idx_h2 + (3) * FUSION_SIZE_SLICE_1_H2) * FUSION_SIZE_SLICE_1_H3];

				reg_singles[1][0] += sm_a[0][idx_p6 + (idx_h1) * FUSION_SIZE_SLICE_1_P6] * sm_b[1][idx_h3 + (idx_h2 + (0) * FUSION_SIZE_SLICE_1_H2) * FUSION_SIZE_SLICE_1_H3];
				reg_singles[1][1] += sm_a[0][idx_p6 + (idx_h1) * FUSION_SIZE_SLICE_1_P6] * sm_b[1][idx_h3 + (idx_h2 + (1) * FUSION_SIZE_SLICE_1_H2) * FUSION_SIZE_SLICE_1_H3];
				reg_singles[1][2] += sm_a[0][idx_p6 + (idx_h1) * FUSION_SIZE_SLICE_1_P6] * sm_b[1][idx_h3 + (idx_h2 + (2) * FUSION_SIZE_SLICE_1_H2) * FUSION_SIZE_SLICE_1_H3];
				reg_singles[1][3] += sm_a[0][idx_p6 + (idx_h1) * FUSION_SIZE_SLICE_1_P6] * sm_b[1][idx_h3 + (idx_h2 + (3) * FUSION_SIZE_SLICE_1_H2) * FUSION_SIZE_SLICE_1_H3];

				reg_singles[2][0] += sm_a[0][idx_p6 + (idx_h1) * FUSION_SIZE_SLICE_1_P6] * sm_b[2][idx_h3 + (idx_h2 + (0) * FUSION_SIZE_SLICE_1_H2) * FUSION_SIZE_SLICE_1_H3];
				reg_singles[2][1] += sm_a[0][idx_p6 + (idx_h1) * FUSION_SIZE_SLICE_1_P6] * sm_b[2][idx_h3 + (idx_h2 + (1) * FUSION_SIZE_SLICE_1_H2) * FUSION_SIZE_SLICE_1_H3];
				reg_singles[2][2] += sm_a[0][idx_p6 + (idx_h1) * FUSION_SIZE_SLICE_1_P6] * sm_b[2][idx_h3 + (idx_h2 + (2) * FUSION_SIZE_SLICE_1_H2) * FUSION_SIZE_SLICE_1_H3];
				reg_singles[2][3] += sm_a[0][idx_p6 + (idx_h1) * FUSION_SIZE_SLICE_1_P6] * sm_b[2][idx_h3 + (idx_h2 + (3) * FUSION_SIZE_SLICE_1_H2) * FUSION_SIZE_SLICE_1_H3];

				reg_singles[3][0] += sm_a[0][idx_p6 + (idx_h1) * FUSION_SIZE_SLICE_1_P6] * sm_b[3][idx_h3 + (idx_h2 + (0) * FUSION_SIZE_SLICE_1_H2) * FUSION_SIZE_SLICE_1_H3];
				reg_singles[3][1] += sm_a[0][idx_p6 + (idx_h1) * FUSION_SIZE_SLICE_1_P6] * sm_b[3][idx_h3 + (idx_h2 + (1) * FUSION_SIZE_SLICE_1_H2) * FUSION_SIZE_SLICE_1_H3];
				reg_singles[3][2] += sm_a[0][idx_p6 + (idx_h1) * FUSION_SIZE_SLICE_1_P6] * sm_b[3][idx_h3 + (idx_h2 + (2) * FUSION_SIZE_SLICE_1_H2) * FUSION_SIZE_SLICE_1_H3];
				reg_singles[3][3] += sm_a[0][idx_p6 + (idx_h1) * FUSION_SIZE_SLICE_1_P6] * sm_b[3][idx_h3 + (idx_h2 + (3) * FUSION_SIZE_SLICE_1_H2) * FUSION_SIZE_SLICE_1_H3];
				__syncthreads();
			}
			
			//
			//  >> s1_8:   t3[h3,h2,h1,p6,p5,p4] -= t2[p6,h2] * v2[h3,h1,p5,p4] (h3,h2,p6), (h1)
			//
			if (flag_s1_8 >= 0)	// these if-conditions make 100 ms..
			{
				// 
				double* tmp_dev_s1_t2_8 = dev_s1_t2_all + size_max_dim_s1_t2 * flag_s1_8;
				double* tmp_dev_s1_v2_8 = dev_s1_v2_all + size_max_dim_s1_v2 * flag_s1_8;

				if (idx_h3 < rng_p6 && idx_h2 < rng_h2 && idx_p6 == 0 && idx_h1 == 0)
				sm_a[0][idx_h3 + (idx_h2) * FUSION_SIZE_SLICE_1_P6] = tmp_dev_s1_t2_8[str_blk_idx_p6 + idx_h3 + (str_blk_idx_h2 + idx_h2) * base_size_p6b];
						
				if (idx_h3 < rng_h3 && idx_h2 < rng_h1 && idx_p6 < rng_p5 && idx_h1 < rng_p4)
				sm_b[idx_h1][idx_h3 + (idx_h2 + (idx_p6) * FUSION_SIZE_SLICE_1_H1) * FUSION_SIZE_SLICE_1_H3] = tmp_dev_s1_v2_8[str_blk_idx_h3 + idx_h3 + (str_blk_idx_h1 + idx_h2 + (str_blk_idx_p5 + idx_p6 + (str_blk_idx_p4 + idx_h1) * base_size_p5b) * base_size_h1b) * base_size_h3b];
				__syncthreads();

				//  "p4" x "p5"
				reg_singles[0][0] -= sm_a[0][idx_p6 + (idx_h2) * FUSION_SIZE_SLICE_1_P6] * sm_b[0][idx_h3 + (idx_h1 + (0) * FUSION_SIZE_SLICE_1_H1) * FUSION_SIZE_SLICE_1_H3];
				reg_singles[0][1] -= sm_a[0][idx_p6 + (idx_h2) * FUSION_SIZE_SLICE_1_P6] * sm_b[0][idx_h3 + (idx_h1 + (1) * FUSION_SIZE_SLICE_1_H1) * FUSION_SIZE_SLICE_1_H3];
				reg_singles[0][2] -= sm_a[0][idx_p6 + (idx_h2) * FUSION_SIZE_SLICE_1_P6] * sm_b[0][idx_h3 + (idx_h1 + (2) * FUSION_SIZE_SLICE_1_H1) * FUSION_SIZE_SLICE_1_H3];
				reg_singles[0][3] -= sm_a[0][idx_p6 + (idx_h2) * FUSION_SIZE_SLICE_1_P6] * sm_b[0][idx_h3 + (idx_h1 + (3) * FUSION_SIZE_SLICE_1_H1) * FUSION_SIZE_SLICE_1_H3];

				reg_singles[1][0] -= sm_a[0][idx_p6 + (idx_h2) * FUSION_SIZE_SLICE_1_P6] * sm_b[1][idx_h3 + (idx_h1 + (0) * FUSION_SIZE_SLICE_1_H1) * FUSION_SIZE_SLICE_1_H3];
				reg_singles[1][1] -= sm_a[0][idx_p6 + (idx_h2) * FUSION_SIZE_SLICE_1_P6] * sm_b[1][idx_h3 + (idx_h1 + (1) * FUSION_SIZE_SLICE_1_H1) * FUSION_SIZE_SLICE_1_H3];
				reg_singles[1][2] -= sm_a[0][idx_p6 + (idx_h2) * FUSION_SIZE_SLICE_1_P6] * sm_b[1][idx_h3 + (idx_h1 + (2) * FUSION_SIZE_SLICE_1_H1) * FUSION_SIZE_SLICE_1_H3];
				reg_singles[1][3] -= sm_a[0][idx_p6 + (idx_h2) * FUSION_SIZE_SLICE_1_P6] * sm_b[1][idx_h3 + (idx_h1 + (3) * FUSION_SIZE_SLICE_1_H1) * FUSION_SIZE_SLICE_1_H3];

				reg_singles[2][0] -= sm_a[0][idx_p6 + (idx_h2) * FUSION_SIZE_SLICE_1_P6] * sm_b[2][idx_h3 + (idx_h1 + (0) * FUSION_SIZE_SLICE_1_H1) * FUSION_SIZE_SLICE_1_H3];
				reg_singles[2][1] -= sm_a[0][idx_p6 + (idx_h2) * FUSION_SIZE_SLICE_1_P6] * sm_b[2][idx_h3 + (idx_h1 + (1) * FUSION_SIZE_SLICE_1_H1) * FUSION_SIZE_SLICE_1_H3];
				reg_singles[2][2] -= sm_a[0][idx_p6 + (idx_h2) * FUSION_SIZE_SLICE_1_P6] * sm_b[2][idx_h3 + (idx_h1 + (2) * FUSION_SIZE_SLICE_1_H1) * FUSION_SIZE_SLICE_1_H3];
				reg_singles[2][3] -= sm_a[0][idx_p6 + (idx_h2) * FUSION_SIZE_SLICE_1_P6] * sm_b[2][idx_h3 + (idx_h1 + (3) * FUSION_SIZE_SLICE_1_H1) * FUSION_SIZE_SLICE_1_H3];

				reg_singles[3][0] -= sm_a[0][idx_p6 + (idx_h2) * FUSION_SIZE_SLICE_1_P6] * sm_b[3][idx_h3 + (idx_h1 + (0) * FUSION_SIZE_SLICE_1_H1) * FUSION_SIZE_SLICE_1_H3];
				reg_singles[3][1] -= sm_a[0][idx_p6 + (idx_h2) * FUSION_SIZE_SLICE_1_P6] * sm_b[3][idx_h3 + (idx_h1 + (1) * FUSION_SIZE_SLICE_1_H1) * FUSION_SIZE_SLICE_1_H3];
				reg_singles[3][2] -= sm_a[0][idx_p6 + (idx_h2) * FUSION_SIZE_SLICE_1_P6] * sm_b[3][idx_h3 + (idx_h1 + (2) * FUSION_SIZE_SLICE_1_H1) * FUSION_SIZE_SLICE_1_H3];
				reg_singles[3][3] -= sm_a[0][idx_p6 + (idx_h2) * FUSION_SIZE_SLICE_1_P6] * sm_b[3][idx_h3 + (idx_h1 + (3) * FUSION_SIZE_SLICE_1_H1) * FUSION_SIZE_SLICE_1_H3];
				__syncthreads();
			}

			//
			//  >> s1_9:   t3[h3,h2,h1,p6,p5,p4] -= t2[p6,h3] * v2[h2,h1,p5,p4] (h3,h2,p6), (h1)
			//
			if (flag_s1_9 >= 0)	// these if-conditions make 100 ms..
			{
				// 
				double* tmp_dev_s1_t2_9 = dev_s1_t2_all + size_max_dim_s1_t2 * flag_s1_9;
				double* tmp_dev_s1_v2_9 = dev_s1_v2_all + size_max_dim_s1_v2 * flag_s1_9;

				if (idx_h3 < rng_p6 && idx_h2 < rng_h3 && idx_p6 == 0 && idx_h1 == 0)
				sm_a[0][idx_h3 + (idx_h2) * FUSION_SIZE_SLICE_1_P6] = tmp_dev_s1_t2_9[str_blk_idx_p6 + idx_h3 + (str_blk_idx_h3 + idx_h2) * base_size_p6b];

				if (idx_h3 < rng_h2 && idx_h2 < rng_h1 && idx_p6 < rng_p5 && idx_h1 < rng_p4)
				sm_b[idx_h1][idx_h3 + (idx_h2 + (idx_p6) * FUSION_SIZE_SLICE_1_H1) * FUSION_SIZE_SLICE_1_H2] = tmp_dev_s1_v2_9[str_blk_idx_h2 + idx_h3 + (str_blk_idx_h1 + idx_h2 + (str_blk_idx_p5 + idx_p6 + (str_blk_idx_p4 + idx_h1) * base_size_p5b) * base_size_h1b) * base_size_h2b];
				__syncthreads();
				
				//  "p4" x "p5"
				reg_singles[0][0] += sm_a[0][idx_p6 + (idx_h3) * FUSION_SIZE_SLICE_1_P6] * sm_b[0][idx_h2 + (idx_h1 + (0) * FUSION_SIZE_SLICE_1_H1) * FUSION_SIZE_SLICE_1_H2];
				reg_singles[0][1] += sm_a[0][idx_p6 + (idx_h3) * FUSION_SIZE_SLICE_1_P6] * sm_b[0][idx_h2 + (idx_h1 + (1) * FUSION_SIZE_SLICE_1_H1) * FUSION_SIZE_SLICE_1_H2];
				reg_singles[0][2] += sm_a[0][idx_p6 + (idx_h3) * FUSION_SIZE_SLICE_1_P6] * sm_b[0][idx_h2 + (idx_h1 + (2) * FUSION_SIZE_SLICE_1_H1) * FUSION_SIZE_SLICE_1_H2];
				reg_singles[0][3] += sm_a[0][idx_p6 + (idx_h3) * FUSION_SIZE_SLICE_1_P6] * sm_b[0][idx_h2 + (idx_h1 + (3) * FUSION_SIZE_SLICE_1_H1) * FUSION_SIZE_SLICE_1_H2];

				reg_singles[1][0] += sm_a[0][idx_p6 + (idx_h3) * FUSION_SIZE_SLICE_1_P6] * sm_b[1][idx_h2 + (idx_h1 + (0) * FUSION_SIZE_SLICE_1_H1) * FUSION_SIZE_SLICE_1_H2];
				reg_singles[1][1] += sm_a[0][idx_p6 + (idx_h3) * FUSION_SIZE_SLICE_1_P6] * sm_b[1][idx_h2 + (idx_h1 + (1) * FUSION_SIZE_SLICE_1_H1) * FUSION_SIZE_SLICE_1_H2];
				reg_singles[1][2] += sm_a[0][idx_p6 + (idx_h3) * FUSION_SIZE_SLICE_1_P6] * sm_b[1][idx_h2 + (idx_h1 + (2) * FUSION_SIZE_SLICE_1_H1) * FUSION_SIZE_SLICE_1_H2];
				reg_singles[1][3] += sm_a[0][idx_p6 + (idx_h3) * FUSION_SIZE_SLICE_1_P6] * sm_b[1][idx_h2 + (idx_h1 + (3) * FUSION_SIZE_SLICE_1_H1) * FUSION_SIZE_SLICE_1_H2];

				reg_singles[2][0] += sm_a[0][idx_p6 + (idx_h3) * FUSION_SIZE_SLICE_1_P6] * sm_b[2][idx_h2 + (idx_h1 + (0) * FUSION_SIZE_SLICE_1_H1) * FUSION_SIZE_SLICE_1_H2];
				reg_singles[2][1] += sm_a[0][idx_p6 + (idx_h3) * FUSION_SIZE_SLICE_1_P6] * sm_b[2][idx_h2 + (idx_h1 + (1) * FUSION_SIZE_SLICE_1_H1) * FUSION_SIZE_SLICE_1_H2];
				reg_singles[2][2] += sm_a[0][idx_p6 + (idx_h3) * FUSION_SIZE_SLICE_1_P6] * sm_b[2][idx_h2 + (idx_h1 + (2) * FUSION_SIZE_SLICE_1_H1) * FUSION_SIZE_SLICE_1_H2];
				reg_singles[2][3] += sm_a[0][idx_p6 + (idx_h3) * FUSION_SIZE_SLICE_1_P6] * sm_b[2][idx_h2 + (idx_h1 + (3) * FUSION_SIZE_SLICE_1_H1) * FUSION_SIZE_SLICE_1_H2];

				reg_singles[3][0] += sm_a[0][idx_p6 + (idx_h3) * FUSION_SIZE_SLICE_1_P6] * sm_b[3][idx_h2 + (idx_h1 + (0) * FUSION_SIZE_SLICE_1_H1) * FUSION_SIZE_SLICE_1_H2];
				reg_singles[3][1] += sm_a[0][idx_p6 + (idx_h3) * FUSION_SIZE_SLICE_1_P6] * sm_b[3][idx_h2 + (idx_h1 + (1) * FUSION_SIZE_SLICE_1_H1) * FUSION_SIZE_SLICE_1_H2];
				reg_singles[3][2] += sm_a[0][idx_p6 + (idx_h3) * FUSION_SIZE_SLICE_1_P6] * sm_b[3][idx_h2 + (idx_h1 + (2) * FUSION_SIZE_SLICE_1_H1) * FUSION_SIZE_SLICE_1_H2];
				reg_singles[3][3] += sm_a[0][idx_p6 + (idx_h3) * FUSION_SIZE_SLICE_1_P6] * sm_b[3][idx_h2 + (idx_h1 + (3) * FUSION_SIZE_SLICE_1_H1) * FUSION_SIZE_SLICE_1_H2];
				__syncthreads();
			}
		}
	}
	
	//
	//  energies
	// 
	double energy_1 = 0.0;
	double energy_2 = 0.0;

	// 
	if (idx_h3 < energy_rng_h3 && idx_h2 < energy_rng_h2 && idx_p6 < energy_rng_p6 && idx_h1 < energy_rng_h1)
	{
		for (int i = 0; i < FUSION_SIZE_SLICE_1_P5; i++)
		{
			for (int j = 0; j < FUSION_SIZE_SLICE_1_P4; j++)
			{
				if (i < energy_rng_p5 && j < energy_rng_p4)
				{
					// 
					double inner_factor = partial_inner_factor - dev_evl_sorted_p5b[i + (energy_str_blk_idx_p5)] - dev_evl_sorted_p4b[j + (energy_str_blk_idx_p4)];

					// 
					energy_1 += (reg_tile[j][i] *  reg_tile[j][i]) 						/ inner_factor;
					energy_2 += (reg_tile[j][i] * (reg_tile[j][i] + reg_singles[j][i])) / inner_factor;
				}
			}
		}
	}
	__syncthreads();
	
	// 
	//  to partially reduce the energies--- E(4) and E(5)
	//  a warp: 32 -(1)-> 16 -(2)-> 8 -(3)-> 4 -(4)-> 2 
	// 
	for (int offset = 16; offset > 0; offset /= 2)
	{
		energy_1 += __shfl_down_sync(FULL_MASK, energy_1, offset);
		energy_2 += __shfl_down_sync(FULL_MASK, energy_2, offset);
	}

	if (threadIdx.x == 0 && threadIdx.y % 2 == 0)
	{
		sm_a[0][threadIdx.y / 2] = energy_1;
		sm_b[0][threadIdx.y / 2] = energy_2;
	}
	__syncthreads();

	// 
	double final_energy_1 = 0.0;
	double final_energy_2 = 0.0;
	if (threadIdx.x == 0 && threadIdx.y == 0)
	{
		for (int i = 0; i < 8; i++)
		{
			final_energy_1 += sm_a[0][i];
			final_energy_2 += sm_b[0][i];
		}

		reduced_energy[blockIdx.x]              = final_energy_1;
		reduced_energy[blockIdx.x + gridDim.x]  = final_energy_2;
	}
}

// 
void total_fused_ccsd_t(size_t base_size_h1b, size_t base_size_h2b, size_t base_size_h3b, 
						size_t base_size_p4b, size_t base_size_p5b, size_t base_size_p6b,
						// 
						double* host_d1_t2_all, double* host_d1_v2_all,
						double* host_d2_t2_all, double* host_d2_v2_all,
						double* host_s1_t2_all, double* host_s1_v2_all,
						// 
						size_t size_d1_t2_all, size_t size_d1_v2_all,
						size_t size_d2_t2_all, size_t size_d2_v2_all,
						size_t size_s1_t2_all, size_t size_s1_v2_all,
						// 
						size_t* list_d1_sizes, 
						size_t* list_d2_sizes, 
						size_t* list_s1_sizes, 
						// 
						std::vector<int> vec_d1_flags,
						std::vector<int> vec_d2_flags,
						std::vector<int> vec_s1_flags, std::vector<int> vec_s1_ai6, 
						// 
						size_t size_noab, size_t size_max_dim_d1_t2, size_t size_max_dim_d1_v2,
						size_t size_nvab, size_t size_max_dim_d2_t2, size_t size_max_dim_d2_v2,
                                          size_t size_max_dim_s1_t2, size_t size_max_dim_s1_v2, 
						// 
						double factor, 
						double* host_evl_sorted_h1b, double* host_evl_sorted_h2b, double* host_evl_sorted_h3b, 
						double* host_evl_sorted_p4b, double* host_evl_sorted_p5b, double* host_evl_sorted_p6b,
						double* final_energy_4, double* final_energy_5)
{
	// 
	double* dev_d1_t2_all = (double*)getGpuMem(size_d1_t2_all * sizeof(double));
	double* dev_d1_v2_all = (double*)getGpuMem(size_d1_v2_all * sizeof(double));
	double* dev_d2_t2_all = (double*)getGpuMem(size_d2_t2_all * sizeof(double));
	double* dev_d2_v2_all = (double*)getGpuMem(size_d2_v2_all * sizeof(double));
	double* dev_s1_t2_all = (double*)getGpuMem(size_s1_t2_all * sizeof(double));
	double* dev_s1_v2_all = (double*)getGpuMem(size_s1_v2_all * sizeof(double));

	double* dev_evl_sorted_h1b = (double*)getGpuMem(base_size_h1b * sizeof(double));
	double* dev_evl_sorted_h2b = (double*)getGpuMem(base_size_h2b * sizeof(double));
	double* dev_evl_sorted_h3b = (double*)getGpuMem(base_size_h3b * sizeof(double));
	double* dev_evl_sorted_p4b = (double*)getGpuMem(base_size_p4b * sizeof(double));
	double* dev_evl_sorted_p5b = (double*)getGpuMem(base_size_p5b * sizeof(double));
	double* dev_evl_sorted_p6b = (double*)getGpuMem(base_size_p6b * sizeof(double));

	// 
	hipMemcpyToSymbol(HIP_SYMBOL(const_list_s1_flags_offset), &vec_s1_flags[0], sizeof(int) * (NUM_IA6_LOOPS * NUM_S1_EQUATIONS));
	hipMemcpyToSymbol(HIP_SYMBOL(const_list_d1_flags_offset), &vec_d1_flags[0], sizeof(int) * (NUM_IA6_LOOPS * MAX_NOAB * NUM_D1_EQUATIONS));
	hipMemcpyToSymbol(HIP_SYMBOL(const_list_d2_flags_offset), &vec_d2_flags[0], sizeof(int) * (NUM_IA6_LOOPS * MAX_NVAB * NUM_D2_EQUATIONS));

	// 
	int tmp_list_s1_sizes[NUM_IA6_LOOPS * NUM_S1_INDEX];
	for (int i = 0; i < NUM_IA6_LOOPS; i++)
	{
		tmp_list_s1_sizes[0 + (i) * NUM_S1_INDEX] = (int)list_s1_sizes[0 + (i) * NUM_S1_INDEX];
		tmp_list_s1_sizes[1 + (i) * NUM_S1_INDEX] = (int)list_s1_sizes[1 + (i) * NUM_S1_INDEX];
		tmp_list_s1_sizes[2 + (i) * NUM_S1_INDEX] = (int)list_s1_sizes[2 + (i) * NUM_S1_INDEX];
		tmp_list_s1_sizes[3 + (i) * NUM_S1_INDEX] = (int)list_s1_sizes[3 + (i) * NUM_S1_INDEX];
		tmp_list_s1_sizes[4 + (i) * NUM_S1_INDEX] = (int)list_s1_sizes[4 + (i) * NUM_S1_INDEX];
		tmp_list_s1_sizes[5 + (i) * NUM_S1_INDEX] = (int)list_s1_sizes[5 + (i) * NUM_S1_INDEX];
	}

	int tmp_list_d1_sizes[NUM_IA6_LOOPS * size_noab * NUM_D1_INDEX];
	for (int i = 0; i < NUM_IA6_LOOPS; i++)
	{
		for (int j = 0; j < size_noab; j++)
		{
			tmp_list_d1_sizes[0 + (j + (i) * size_noab) * NUM_D1_INDEX] = (int)list_d1_sizes[0 + (j + (i) * size_noab) * NUM_D1_INDEX];
			tmp_list_d1_sizes[1 + (j + (i) * size_noab) * NUM_D1_INDEX] = (int)list_d1_sizes[1 + (j + (i) * size_noab) * NUM_D1_INDEX];
			tmp_list_d1_sizes[2 + (j + (i) * size_noab) * NUM_D1_INDEX] = (int)list_d1_sizes[2 + (j + (i) * size_noab) * NUM_D1_INDEX];
			tmp_list_d1_sizes[3 + (j + (i) * size_noab) * NUM_D1_INDEX] = (int)list_d1_sizes[3 + (j + (i) * size_noab) * NUM_D1_INDEX];
			tmp_list_d1_sizes[4 + (j + (i) * size_noab) * NUM_D1_INDEX] = (int)list_d1_sizes[4 + (j + (i) * size_noab) * NUM_D1_INDEX];
			tmp_list_d1_sizes[5 + (j + (i) * size_noab) * NUM_D1_INDEX] = (int)list_d1_sizes[5 + (j + (i) * size_noab) * NUM_D1_INDEX];
			tmp_list_d1_sizes[6 + (j + (i) * size_noab) * NUM_D1_INDEX] = (int)list_d1_sizes[6 + (j + (i) * size_noab) * NUM_D1_INDEX];
		}
	}

	int tmp_list_d2_sizes[NUM_IA6_LOOPS * size_nvab * NUM_D2_INDEX];
	for (int i = 0; i < NUM_IA6_LOOPS; i++)
	{
		for (int j = 0; j < size_nvab; j++)
		{
			tmp_list_d2_sizes[0 + (j + (i) * size_nvab) * NUM_D2_INDEX] = (int)list_d2_sizes[0 + (j + (i) * size_nvab) * NUM_D2_INDEX];
			tmp_list_d2_sizes[1 + (j + (i) * size_nvab) * NUM_D2_INDEX] = (int)list_d2_sizes[1 + (j + (i) * size_nvab) * NUM_D2_INDEX];
			tmp_list_d2_sizes[2 + (j + (i) * size_nvab) * NUM_D2_INDEX] = (int)list_d2_sizes[2 + (j + (i) * size_nvab) * NUM_D2_INDEX];
			tmp_list_d2_sizes[3 + (j + (i) * size_nvab) * NUM_D2_INDEX] = (int)list_d2_sizes[3 + (j + (i) * size_nvab) * NUM_D2_INDEX];
			tmp_list_d2_sizes[4 + (j + (i) * size_nvab) * NUM_D2_INDEX] = (int)list_d2_sizes[4 + (j + (i) * size_nvab) * NUM_D2_INDEX];
			tmp_list_d2_sizes[5 + (j + (i) * size_nvab) * NUM_D2_INDEX] = (int)list_d2_sizes[5 + (j + (i) * size_nvab) * NUM_D2_INDEX];
			tmp_list_d2_sizes[6 + (j + (i) * size_nvab) * NUM_D2_INDEX] = (int)list_d2_sizes[6 + (j + (i) * size_nvab) * NUM_D2_INDEX];
		}
	}

	//
	hipMemcpyToSymbol(HIP_SYMBOL(const_list_s1_problem_size), tmp_list_s1_sizes, sizeof(int) * (NUM_IA6_LOOPS * NUM_S1_INDEX));
	hipMemcpyToSymbol(HIP_SYMBOL(const_list_d1_problem_size), tmp_list_d1_sizes, sizeof(int) * (NUM_IA6_LOOPS * NUM_D1_INDEX * size_noab));
	hipMemcpyToSymbol(HIP_SYMBOL(const_list_d2_problem_size), tmp_list_d2_sizes, sizeof(int) * (NUM_IA6_LOOPS * NUM_D2_INDEX * size_nvab));

	// 
	hipMemcpy(dev_d1_t2_all, host_d1_t2_all, (size_d1_t2_all) * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(dev_d1_v2_all, host_d1_v2_all, (size_d1_v2_all) * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(dev_d2_t2_all, host_d2_t2_all, (size_d2_t2_all) * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(dev_d2_v2_all, host_d2_v2_all, (size_d2_v2_all) * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(dev_s1_t2_all, host_s1_t2_all, (size_s1_t2_all) * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(dev_s1_v2_all, host_s1_v2_all, (size_s1_v2_all) * sizeof(double), hipMemcpyHostToDevice);

	// 
	hipMemcpy(dev_evl_sorted_h1b, host_evl_sorted_h1b, base_size_h1b * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(dev_evl_sorted_h2b, host_evl_sorted_h2b, base_size_h2b * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(dev_evl_sorted_h3b, host_evl_sorted_h3b, base_size_h3b * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(dev_evl_sorted_p4b, host_evl_sorted_p4b, base_size_p4b * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(dev_evl_sorted_p5b, host_evl_sorted_p5b, base_size_p5b * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(dev_evl_sorted_p6b, host_evl_sorted_p6b, base_size_p6b * sizeof(double), hipMemcpyHostToDevice);
	
	//
	//  the kernel should be based on the based problem sizes.
	// 
	size_t num_blocks_kernel_1 = CEIL(base_size_h3b, FUSION_SIZE_SLICE_1_H3) * CEIL(base_size_h2b, FUSION_SIZE_SLICE_1_H2) * 
								 CEIL(base_size_h1b, FUSION_SIZE_SLICE_1_H1) * CEIL(base_size_p6b, FUSION_SIZE_SLICE_1_P6) * 
								 CEIL(base_size_p5b, FUSION_SIZE_SLICE_1_P5) * CEIL(base_size_p4b, FUSION_SIZE_SLICE_1_P4);
	
	// Depends on # of Fused Kernel
	dim3 gridsize_1(num_blocks_kernel_1);
	dim3 blocksize_1(FUSION_SIZE_TB_1_X, FUSION_SIZE_TB_1_Y);

	// 
	double* host_energies = (double*)malloc(num_blocks_kernel_1 * NUM_ENERGIES * sizeof(double));
	// memset(host_energies, 0.0, num_blocks_kernel_1 * NUM_ENERGIES * sizeof(double));
	double* dev_energies = (double*)getGpuMem(num_blocks_kernel_1 * NUM_ENERGIES * sizeof(double));

#ifdef DEBUG_TIME_FUSED_CCSD_T
	hipEvent_t start_ccsd_t, stop_ccsd_t, stop_kernely_only;
	hipEventCreate(&start_ccsd_t);
	hipEventCreate(&stop_ccsd_t);
	hipEventCreate(&stop_kernely_only);
	hipEventRecord(start_ccsd_t);
#endif

	// 
	jk_ccsd_t_fully_fused_kernel<<<gridsize_1, blocksize_1>>>((int)size_noab, (int)size_nvab, 
																// 
																(int)size_max_dim_s1_t2, (int)size_max_dim_s1_v2,
																(int)size_max_dim_d1_t2, (int)size_max_dim_d1_v2,
																(int)size_max_dim_d2_t2, (int)size_max_dim_d2_v2,
																// 
																dev_d1_t2_all, dev_d1_v2_all, 
																dev_d2_t2_all, dev_d2_v2_all, 
																dev_s1_t2_all, dev_s1_v2_all, 
																//  
																dev_evl_sorted_h1b, dev_evl_sorted_h2b, dev_evl_sorted_h3b,
																dev_evl_sorted_p4b, dev_evl_sorted_p5b, dev_evl_sorted_p6b,
																dev_energies, 
																// 
																CEIL(base_size_h3b, FUSION_SIZE_SLICE_1_H3), CEIL(base_size_h2b, FUSION_SIZE_SLICE_1_H2), CEIL(base_size_h1b, FUSION_SIZE_SLICE_1_H1), 
																CEIL(base_size_p6b, FUSION_SIZE_SLICE_1_P6), CEIL(base_size_p5b, FUSION_SIZE_SLICE_1_P5), CEIL(base_size_p4b, FUSION_SIZE_SLICE_1_P4),
																// 
																(int)base_size_h1b, (int)base_size_h2b, (int)base_size_h1b, 
																(int)base_size_p4b, (int)base_size_p5b, (int)base_size_p6b);

	//
#ifdef DEBUG_TIME_FUSED_CCSD_T
	hipEventRecord(stop_kernely_only);
	hipEventSynchronize(stop_kernely_only);
	float time_ms_ccsd_t_kernel_only = 0.0;
	hipEventElapsedTime(&time_ms_ccsd_t_kernel_only, start_ccsd_t, stop_kernely_only);
#endif
	
	//
	hipMemcpy(host_energies, dev_energies, num_blocks_kernel_1 * NUM_ENERGIES * sizeof(double), hipMemcpyDeviceToHost);

	// 
	double final_energy_1 = 0.0;
	double final_energy_2 = 0.0;
	for (size_t i = 0; i < num_blocks_kernel_1; i++)
	{
		// 
		final_energy_1 += host_energies[i];
		final_energy_2 += host_energies[i + num_blocks_kernel_1];
	}

	// 
	// final_energy_1 *= factor;
	// final_energy_2 *= factor;
	*final_energy_4 = final_energy_1 * factor;
	*final_energy_5 = final_energy_2 * factor;

#ifdef DEBUG_TIME_FUSED_CCSD_T
	hipEventRecord(stop_ccsd_t);
	hipEventSynchronize(stop_ccsd_t);
	float time_ms_ccsd_t_kernel = 0.0;
	hipEventElapsedTime(&time_ms_ccsd_t_kernel, start_ccsd_t, stop_ccsd_t);
	printf ("========================================================================================\n");
	printf ("[%s][fused] kernel-only-time: %f (ms)\n", __func__, time_ms_ccsd_t_kernel_only);
	printf ("[%s][fused] total-time: %f (ms)\n", __func__, time_ms_ccsd_t_kernel);
	printf ("[%s][fused] E(4): %.15f, E(5): %.15f\n", __func__,  final_energy_1,  final_energy_2);
	printf ("[%s][fused] E(4): %.15f, E(5): %.15f\n", __func__, *final_energy_4, *final_energy_5);
	printf ("========================================================================================\n");
#endif

	//
	freeGpuMem(dev_s1_t2_all);	freeGpuMem(dev_s1_v2_all);
	freeGpuMem(dev_d1_t2_all);	freeGpuMem(dev_d1_v2_all);
	freeGpuMem(dev_d2_t2_all);	freeGpuMem(dev_d2_v2_all);

	freeGpuMem(dev_evl_sorted_h1b); freeGpuMem(dev_evl_sorted_h2b); freeGpuMem(dev_evl_sorted_h3b);
	freeGpuMem(dev_evl_sorted_p4b); freeGpuMem(dev_evl_sorted_p5b); freeGpuMem(dev_evl_sorted_p6b);

	freeGpuMem(dev_energies);
	free(host_energies);
}